
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "PipelineGatherOutput.hpp"

#include "Context.hpp"

/** @brief Gather output kernel. */
__global__ void gatherOutputKernel(
    int32_t* ray_index_buffer,
    ldplab::Vec3* force_per_ray,
    ldplab::Vec3* torque_per_ray,
    size_t num_rays_per_batch,
    ldplab::Vec3* force_per_particle,
    ldplab::Vec3* torque_per_particle,
    ldplab::Mat3* p2w_transformations,
    size_t num_particles,
    bool particle_space_output);

void ldplab::rtscuda::PipelineGatherOutput::execute(size_t ray_buffer_index)
{
    const size_t grid_size = m_context.parameters.num_particles;
    const size_t block_size = m_context.parameters.num_threads_per_block;
    const size_t shared_mem_size = block_size * sizeof(Vec3) * 2;
    gatherOutputKernel<<<grid_size, block_size, shared_mem_size>>>(
        m_context.resources.ray_buffer.index_buffers[ray_buffer_index].get(),
        m_context.resources.output_buffer.force_per_ray.get(),
        m_context.resources.output_buffer.torque_per_ray.get(),
        m_context.parameters.num_rays_per_batch,
        m_context.resources.output_buffer.force_per_particle.get(),
        m_context.resources.output_buffer.torque_per_particle.get(),
        m_context.resources.transformations.p2w_transformation.get(),
        m_context.parameters.num_particles,
        m_context.parameters.output_in_particle_space);
}

__global__ void gatherOutputKernel(
    int32_t* ray_index_buffer,
    ldplab::Vec3* force_per_ray,
    ldplab::Vec3* torque_per_ray,
    size_t num_rays_per_batch,
    ldplab::Vec3* force_per_particle,
    ldplab::Vec3* torque_per_particle,
    ldplab::Mat3* p2w_transformations,
    size_t num_particles,
    bool particle_space_output)
{
    using namespace ldplab;
    using namespace ldplab::rtscuda;

    // Shared memory
    extern __shared__ Vec3 sbuf[];

    // ========================================================================
    // Preparation step: Prepare shared buffer and needed variables
    const unsigned int tid = threadIdx.x;
    const unsigned int pi = blockIdx.x;
    const unsigned int force_idx = tid;
    const unsigned int torque_idx = tid + blockDim.x;
    size_t ray_count = num_rays_per_batch / blockDim.x;
    if (num_rays_per_batch % blockDim.x != 0)
        ++ray_count;
    sbuf[force_idx] = Vec3(0, 0, 0);
    sbuf[torque_idx] = Vec3(0, 0, 0);
    unsigned int ri = tid;
    for (size_t i = 0; i <= ray_count; ++i)
    {
        if (ri < num_rays_per_batch)
        {
            if (ray_index_buffer[ri] == static_cast<int32_t>(pi))
            {
                sbuf[force_idx] += force_per_ray[ri];
                sbuf[torque_idx] += torque_per_ray[ri];
            }
        }
        ri += blockDim.x;
    }
    __syncthreads();

    // ========================================================================
    // Reduce step: Loop over the buffer and reduce its content
    for (unsigned int lim = blockDim.x; lim > 1; lim /= 2)
    {
        unsigned int ofs = lim / 2;
        if (tid + ofs < lim)
            sbuf[tid] += sbuf[tid + ofs];
        __syncthreads();
    }
    for (unsigned int lim = blockDim.x; lim > 1; lim /= 2)
    {
        unsigned int ofs = lim / 2;
        if (tid + ofs < lim)
            sbuf[tid + blockDim.x] += sbuf[tid + blockDim.x + ofs];
        __syncthreads();
    }

    // ========================================================================
    // Final step: Write the result from shared buffer in output buffers
    if (tid == 0)
    {
        if (!particle_space_output)
        {
            sbuf[force_idx] = p2w_transformations[pi] * sbuf[force_idx];
            sbuf[torque_idx] = p2w_transformations[pi] * sbuf[torque_idx];
        }
        force_per_particle[pi] += sbuf[force_idx];
        torque_per_particle[pi] += sbuf[torque_idx];
    }
}

#endif