
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "PipelineGatherOutput.hpp"

#include "Context.hpp"

ldplab::rtscuda::KernelLaunchParameter 
    ldplab::rtscuda::PipelineGatherOutput::getLaunchParameter()
{
    KernelLaunchParameter lp;
    lp.block_size.x = 128;
    lp.grid_size.x = m_context.parameters.num_particles;
    lp.shared_memory_size = lp.block_size.x * sizeof(Vec3) * 2;
    return lp;
}

__global__ void gatherOutputKernel(
    int32_t* ray_index_buffer,
    ldplab::Vec3* force_per_ray,
    ldplab::Vec3* torque_per_ray,
    size_t num_rays_per_batch,
    ldplab::Vec3* force_per_particle,
    ldplab::Vec3* torque_per_particle,
    ldplab::Mat3* p2w_transformations,
    size_t num_particles,
    bool particle_space_output)
{
    using namespace ldplab;
    using namespace ldplab::rtscuda;

    // Shared memory
    extern __shared__ Vec3 sbuf[];

    // ========================================================================
    // Preparation step: Prepare shared buffer and needed variables
    const unsigned int tid = threadIdx.x;
    const unsigned int pi = blockIdx.x;
    const unsigned int force_idx = tid;
    const unsigned int torque_idx = tid + blockDim.x;
    size_t ray_count = num_rays_per_batch / blockDim.x;
    if (num_rays_per_batch % blockDim.x != 0)
        ++ray_count;
    sbuf[force_idx] = Vec3(0, 0, 0);
    sbuf[torque_idx] = Vec3(0, 0, 0);
    unsigned int ri = tid;
    for (size_t i = 0; i <= ray_count; ++i)
    {
        if (ri < num_rays_per_batch)
        {
            if (ray_index_buffer[ri] == static_cast<int32_t>(pi))
            {
                sbuf[force_idx] += force_per_ray[ri];
                sbuf[torque_idx] += torque_per_ray[ri];
            }
        }
        ri += blockDim.x;
    }
    __syncthreads();

    // ========================================================================
    // Reduce step: Loop over the buffer and reduce its content
    for (unsigned int lim = blockDim.x; lim > 1; lim /= 2)
    {
        unsigned int ofs = lim / 2;
        if (tid + ofs < lim)
            sbuf[tid] += sbuf[tid + ofs];
        __syncthreads();
    }
    for (unsigned int lim = blockDim.x; lim > 1; lim /= 2)
    {
        unsigned int ofs = lim / 2;
        if (tid + ofs < lim)
            sbuf[tid + blockDim.x] += sbuf[tid + blockDim.x + ofs];
        __syncthreads();
    }

    // ========================================================================
    // Final step: Write the result from shared buffer in output buffers
    if (tid == 0)
    {
        if (!particle_space_output)
        {
            sbuf[force_idx] = p2w_transformations[pi] * sbuf[force_idx];
            sbuf[torque_idx] = p2w_transformations[pi] * sbuf[torque_idx];
        }
        force_per_particle[pi] += sbuf[force_idx];
        torque_per_particle[pi] += sbuf[torque_idx];
    }
}

void ldplab::rtscuda::PipelineGatherOutput::execute(size_t ray_buffer_index)
{
    //const size_t grid_size = m_context.parameters.num_particles;
    //const size_t block_size = m_context.parameters.num_threads_per_block;
    //const size_t shared_memory_size = block_size * sizeof(Vec3) * 2;
    const KernelLaunchParameter lp = getLaunchParameter();
    gatherOutputKernel << <lp.grid_size, lp.block_size, lp.shared_memory_size >> > (
        m_context.resources.ray_buffer.index_buffers[ray_buffer_index].get(),
        m_context.resources.output_buffer.force_per_ray.get(),
        m_context.resources.output_buffer.torque_per_ray.get(),
        m_context.parameters.num_rays_per_batch,
        m_context.resources.output_buffer.force_per_particle.get(),
        m_context.resources.output_buffer.torque_per_particle.get(),
        m_context.resources.transformations.p2w_transformation.get(),
        m_context.parameters.num_particles,
        m_context.parameters.output_in_particle_space);
}

__device__ void ldplab::rtscuda::executeGatherOutputKernel(
    DevicePipelineResources& resources, 
    size_t ray_buffer_index)
{
    const dim3 grid_sz = resources.launch_params.gatherOutput.grid_size;
    const dim3 block_sz = resources.launch_params.gatherOutput.block_size;
    const unsigned int mem_sz = resources.launch_params.gatherOutput.shared_memory_size;
    gatherOutputKernel<<<grid_sz, block_sz, mem_sz>>>(
        resources.ray_buffer.indices[ray_buffer_index],
        resources.output_buffer.force_per_ray,
        resources.output_buffer.torque_per_ray,
        resources.parameters.num_rays_per_batch,
        resources.output_buffer.force_per_particle,
        resources.output_buffer.torque_per_particle,
        resources.transformations.p2w_transformation,
        resources.parameters.num_particles,
        resources.parameters.output_in_particle_space);
}

#endif