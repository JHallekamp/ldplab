
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "PipelineInnerParticlePropagation.hpp"

#include "Context.hpp"
#include "../../Utils/Log.hpp"

namespace rk4_linear_index_gradient_cuda
{
    __global__ void innerParticlePropagationKernel(
        double step_size,
        int32_t* ray_index_buffer,
        ldplab::Vec3* ray_origin_buffer,
        ldplab::Vec3* ray_direction_buffer,
        double* ray_intensity_buffer,
        ldplab::Vec3* intersection_point_buffer,
        ldplab::Vec3* intersection_normal_buffer,
        size_t num_rays_per_batch,
        ldplab::rtscuda::GenericParticleGeometryData* geometry_per_particle,
        ldplab::rtscuda::GenericParticleMaterialData* material_per_particle,
        ldplab::Vec3* particle_center_of_mass,
        ldplab::Vec3* output_force_per_ray,
        ldplab::Vec3* output_torque_per_ray,
        size_t num_particles);
    __device__ void rk4(
        const ldplab::rtscuda::ParticleLinearOneDirectionalMaterial::Data* material,
        const ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::Arg& x,
        const double h,
        ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::Arg& x_new);
    __device__ void executeKernel(
        ldplab::rtscuda::DevicePipelineResources& resources,
        size_t ray_buffer_index);
    __device__ ldplab::rtscuda::pipelineExecuteInnerParticlePropagationStage_t
        execution_kernel_ptr = executeKernel;
    __device__ double rk4_parameter_step_size;
}

std::shared_ptr<ldplab::rtscuda::IPipelineInnerParticlePropagation>
ldplab::rtscuda::IPipelineInnerParticlePropagation::createInstance(
    const RayTracingStepCUDAInfo& info,
    Context& context)
{
    std::shared_ptr<ldplab::rtscuda::IPipelineInnerParticlePropagation> ipp;
    if (info.solver_parameters->type() == IEikonalSolverParameter::Type::rk4)
    {
        ipp = std::make_shared<PipelineInnerParticlePropagationRK4LinearIndexGradient>
            (context, *static_cast<RK4Parameter*>(info.solver_parameters.get()));
    }
    else
    {
        LDPLAB_LOG_ERROR("RTSCUDA context %i: Inner particle propagation "\
            "stage creation failed, unsupported solver type",
            context.uid);
        return nullptr;
    }

    if (!ipp->allocate())
        return nullptr;
    return ipp;
}

ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::
    PipelineInnerParticlePropagationRK4LinearIndexGradient(
        Context& context, 
        RK4Parameter parameter)
    :
    m_context{ context },
    m_parameters{ parameter }
{ }

ldplab::rtscuda::pipelineExecuteInnerParticlePropagationStage_t 
    ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::
        getKernel()
{
    using namespace rk4_linear_index_gradient_cuda;
    // Copy the function pointer to the host
    pipelineExecuteInnerParticlePropagationStage_t kernel = nullptr;
    if (cudaMemcpyFromSymbol(
        &kernel,
        execution_kernel_ptr,
        sizeof(execution_kernel_ptr))
        != cudaSuccess)
        return nullptr;
    return kernel;
}

void ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::
    execute(size_t ray_buffer_index)
{
    using namespace rk4_linear_index_gradient_cuda;
    static const KernelLaunchParameter lp = getLaunchParameter();
    innerParticlePropagationKernel<<<lp.grid_size, lp.block_size>>>(
        m_parameters.step_size,
        m_context.resources.ray_buffer.index_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.origin_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.direction_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.intensity_buffers[ray_buffer_index].get(),
        m_context.resources.intersection_buffer.intersection_point_buffer.get(),
        m_context.resources.intersection_buffer.intersection_normal_buffer.get(),
        m_context.parameters.num_rays_per_batch,
        m_context.resources.particles.geometry_per_particle.get(),
        m_context.resources.particles.material_per_particle.get(),
        m_context.resources.particles.center_of_mass_per_particle.get(),
        m_context.resources.output_buffer.force_per_ray.get(),
        m_context.resources.output_buffer.torque_per_ray.get(),
        m_context.parameters.num_particles);
}

__device__ void rk4_linear_index_gradient_cuda::executeKernel(
    ldplab::rtscuda::DevicePipelineResources& resources,
    size_t ray_buffer_index)
{
    const dim3 grid_sz = resources.launch_params.innerParticlePropagation.grid_size;
    const dim3 block_sz = resources.launch_params.innerParticlePropagation.block_size;
    const unsigned int mem_sz = resources.launch_params.innerParticlePropagation.shared_memory_size;
    innerParticlePropagationKernel<<<grid_sz, block_sz, mem_sz>>>(
        rk4_parameter_step_size,
        resources.ray_buffer.indices[ray_buffer_index],
        resources.ray_buffer.origins[ray_buffer_index],
        resources.ray_buffer.directions[ray_buffer_index],
        resources.ray_buffer.intensities[ray_buffer_index],
        resources.intersection_buffer.points,
        resources.intersection_buffer.normals,
        resources.parameters.num_rays_per_batch,
        resources.particles.geometry_per_particle,
        resources.particles.material_per_particle,
        resources.particles.center_of_mass_per_particle,
        resources.output_buffer.force_per_ray,
        resources.output_buffer.torque_per_ray,
        resources.parameters.num_particles);
}

ldplab::rtscuda::KernelLaunchParameter 
    ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::
        getLaunchParameter()
{
    KernelLaunchParameter p;
    p.block_size.x = 192;
        //std::min(m_context.device_properties.registers_per_block / 96, 
        //    m_context.device_properties.max_num_threads_per_block);
    p.grid_size.x = m_context.parameters.num_rays_per_batch / p.block_size.x +
        (m_context.parameters.num_rays_per_batch % p.block_size.x ? 1 : 0);
    p.shared_memory_size = 0;
    return p;
}

bool ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::allocate()
{
    if (cudaMemcpyToSymbol(
        rk4_linear_index_gradient_cuda::rk4_parameter_step_size,
        &m_parameters.step_size,
        sizeof(rk4_linear_index_gradient_cuda::rk4_parameter_step_size)) != cudaSuccess)
    {
            LDPLAB_LOG_ERROR("RTSCUDA context %i: Inner particle propagation "\
                "stage allocation failed, couldn't upload parameters",
                m_context.uid);
        return false;
    }
    return true;
}

__global__ void rk4_linear_index_gradient_cuda::innerParticlePropagationKernel(
        double step_size,
        int32_t* ray_index_buffer, 
        ldplab::Vec3* ray_origin_buffer, 
        ldplab::Vec3* ray_direction_buffer, 
        double* ray_intensity_buffer, 
        ldplab::Vec3* intersection_point_buffer, 
        ldplab::Vec3* intersection_normal_buffer, 
        size_t num_rays_per_batch, 
        ldplab::rtscuda::GenericParticleGeometryData* geometry_per_particle, 
        ldplab::rtscuda::GenericParticleMaterialData* material_per_particle,
        ldplab::Vec3* particle_center_of_mass,
        ldplab::Vec3* output_force_per_ray, 
        ldplab::Vec3* output_torque_per_ray, 
        size_t num_particles)
{
    using namespace ldplab;
    using namespace rtscuda;
    unsigned int ri = blockIdx.x * blockDim.x + threadIdx.x;
    if (ri >= num_rays_per_batch)
        return;

    int32_t particle_index = ray_index_buffer[ri];
    if (particle_index < 0 ||
        particle_index >= num_particles)
        return;

    const Vec3 ray_origin = ray_origin_buffer[ri];
    Vec3 ray_direction = ray_direction_buffer[ri];
    const double ray_intesity = ray_intensity_buffer[ri];
    void* const particle_geometry = geometry_per_particle[particle_index].data;
    intersectRayParticleGeometryFunction_t intersectRayParticle = 
        geometry_per_particle[particle_index].intersect_ray_particle;
    ParticleLinearOneDirectionalMaterial::Data* material = 
        static_cast<ParticleLinearOneDirectionalMaterial::Data*>(
            material_per_particle[particle_index].data);
    bool intersected = false;
    bool is_inside = false;
    Vec3 inter_point;
    Vec3 inter_normal;
    PipelineInnerParticlePropagationRK4LinearIndexGradient::Arg x{
        ray_direction * material->indexOfRefraction(ray_origin),
        ray_origin 
    };
    PipelineInnerParticlePropagationRK4LinearIndexGradient::Arg x_new{ };
    while (!intersected)
    {
        rk4(material, x, step_size, x_new);
        intersected = GenericParticleFunctionWrapper::intersectSegment(
            intersectRayParticle,
            x.r,
            x_new.r,
            particle_geometry,
            inter_point,
            inter_normal,
            is_inside);
        if (intersected || !is_inside)
        {
            if (!intersected)
            {
                // The following check is neccessary to test if the ray hits in
                // an extreme point (volume so small that it lies within the 
                // epsilon region). If that is the case, we assume the ray 
                // tunnels through the particle.
                bool intersect_outside = false;
                Vec3 t_ip, t_in;
                intersected = GenericParticleFunctionWrapper::intersectRay(
                    intersectRayParticle,
                    ray_origin,
                    ray_direction,
                    particle_geometry,
                    t_ip,
                    t_in,
                    intersect_outside);
                if (!intersected || intersect_outside)
                {
                    // We have found a case, where the ray tunnels through the
                    // Particle. We use the original ray and invalidate the 
                    // surface normal.
                    inter_point = ray_origin;
                    inter_normal = Vec3(0, 0, 0);
                }
                else
                {
                    // We have found a case, where the initial ray is bend 
                    // out of the particle in the initial step due to the 
                    // particle material gradient. In this case we assume that 
                    // it hits in the original intersection point.
                    // We use the previous normal and have to flip it, to 
                    // ensure that we have correct behaviour in the interaction
                    // stage.
                    // To receive the previous normal, we simply perform the
                    // Intersection test again, but this time we reverse the
                    // segment directions. Then we flip the normal.
                    GenericParticleFunctionWrapper::intersectSegment(
                        intersectRayParticle,
                        x_new.r,
                        x.r,
                        particle_geometry,
                        inter_point,
                        inter_normal,
                        is_inside);
                    inter_point = ray_origin;
                    inter_normal = -inter_normal;
                    ray_direction = glm::normalize(x.w);
                }
            }
            else
            {
                ray_direction = glm::normalize(x.w);
                const double nx = material->indexOfRefraction(x.r);
                const double ny = material->indexOfRefraction(inter_point);
                const Vec3 delta_momentum = (nx - ny) * ray_direction;
                const Vec3 r = inter_point - 
                    particle_center_of_mass[particle_index];
                output_force_per_ray[ri] +=
                    ray_intesity * delta_momentum;
                output_torque_per_ray[ri] +=
                    ray_intesity * glm::cross(r, delta_momentum);
            }
            intersected = true;
        }
        else
        {
            const double nx = material->indexOfRefraction(x.r);
            const double ny = material->indexOfRefraction(x_new.r);
            const Vec3 t_old_direction = glm::normalize(x.w);
            const Vec3 t_new_direction = glm::normalize(x_new.w);
            const Vec3 delta_momentum =
                nx * t_old_direction -
                ny * t_new_direction;
            const Vec3 r = x_new.r - particle_center_of_mass[particle_index];
            output_force_per_ray[ri] += 
                ray_intesity * delta_momentum;
            output_torque_per_ray[ri] += 
                ray_intesity * glm::cross(r, delta_momentum);
            x = x_new;
        }
    }
    ray_direction_buffer[ri] = ray_direction;
    intersection_point_buffer[ri] = inter_point;
    intersection_normal_buffer[ri] = inter_normal;
}

__device__ void rk4_linear_index_gradient_cuda::rk4(
    const ldplab::rtscuda::ParticleLinearOneDirectionalMaterial::Data* material, 
    const ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::Arg& x,
    const double h,
    ldplab::rtscuda::PipelineInnerParticlePropagationRK4LinearIndexGradient::Arg& x_new)
{
    using namespace ldplab;
    using namespace rtscuda;
    PipelineInnerParticlePropagationRK4LinearIndexGradient::Arg k[4]{};
    const double beta[4] = { 1.0, 0.5, 0.5, 1.0 };
    const double c[4] = { 1.0, 2.0, 2.0, 1.0 };
    x_new = { {0,0,0}, {0,0,0} };
    for (size_t i = 0; i < 4; ++i)
    {
        PipelineInnerParticlePropagationRK4LinearIndexGradient::Arg x_step = x;
        if (i > 0)
        {
            const double hb = h * beta[i];
            x_step.w += k[i - 1].w * hb;
            x_step.r += k[i - 1].r * hb;
        }
        // eikonal(particle, x_step)
        k[i].w = material->direction * material->gradient;
        const double index_of_refraction =
            1.0 / material->indexOfRefraction(x_step.r);
        k[i].r = x_step.w * index_of_refraction;
        if (c[i] != 0.0)
        {
            x_new.w += k[i].w * c[i];
            x_new.r += k[i].r * c[i];
        }
    }
    x_new.w *= h / 6.0;
    x_new.r *= h / 6.0;
    x_new.w += x.w;
    x_new.r += x.r;
}

#endif