
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "PipelineBoundingVolumeIntersection.hpp"

#include "Context.hpp"

std::shared_ptr<ldplab::rtscuda::IPipelineBoundingVolumeIntersectionStage> 
    ldplab::rtscuda::IPipelineBoundingVolumeIntersectionStage::createInstance(
        const RayTracingStepCUDAInfo& info, Context& context)
{
    // Currently just create the bruteforce stage
    return std::make_shared<PipelineBoundingVolumeIntersectionBruteforce>(
        context);
}

namespace bruteforce_cuda
{
    /** @brief Bounding volume intersection device kernel. */
    __global__ void bvIntersectionKernel(
        int32_t* ray_index_buffer,
        ldplab::Vec3* ray_origin_buffer,
        ldplab::Vec3* ray_direction_buffer,
        double* ray_min_bv_dist_buffer,
        size_t num_rays_per_batch,
        ldplab::rtscuda::GenericBoundingVolumeData* bounding_volumes,
        ldplab::Mat3* w2p_transformation,
        ldplab::Vec3* w2p_translation,
        size_t num_particles);
    /** @brief Device function pointer to the actual kernel. */
    __device__ ldplab::rtscuda::pipelineBoundingVolumeIntersectionStageKernel_t
        bv_intersection_kernel_ptr = bvIntersectionKernel;
}

ldplab::rtscuda::PipelineBoundingVolumeIntersectionBruteforce::
    PipelineBoundingVolumeIntersectionBruteforce(Context& context)
    :
    m_context{ context }
{ }

void ldplab::rtscuda::PipelineBoundingVolumeIntersectionBruteforce::execute(
    size_t ray_buffer_index)
{
    const size_t block_size = m_context.parameters.num_threads_per_block;
    const size_t grid_size = m_context.parameters.num_rays_per_batch / block_size;
    bruteforce_cuda::bvIntersectionKernel<<<grid_size, block_size>>>(
        m_context.resources.ray_buffer.index_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.origin_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.direction_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.min_bv_dist_buffers[ray_buffer_index].get(),
        m_context.parameters.num_rays_per_batch,
        m_context.resources.bounding_volumes.bounding_volume_per_particle.get(),
        m_context.resources.transformations.w2p_transformation.get(),
        m_context.resources.transformations.w2p_translation.get(),
        m_context.parameters.num_particles);
}

ldplab::rtscuda::pipelineBoundingVolumeIntersectionStageKernel_t 
    ldplab::rtscuda::PipelineBoundingVolumeIntersectionBruteforce::getKernel()
{
    // Copy the function pointer to the host
    pipelineBoundingVolumeIntersectionStageKernel_t kernel = nullptr;
    if (cudaMemcpyFromSymbol(
        &kernel,
        bruteforce_cuda::bv_intersection_kernel_ptr,
        sizeof(bruteforce_cuda::bv_intersection_kernel_ptr))
        != cudaSuccess)
        return nullptr;
    return kernel;
}

__global__ void bruteforce_cuda::bvIntersectionKernel(
        int32_t* ray_index_buffer, 
        ldplab::Vec3* ray_origin_buffer, 
        ldplab::Vec3* ray_direction_buffer, 
        double* ray_min_bv_dist_buffer, 
        size_t num_rays_per_batch,
        ldplab::rtscuda::GenericBoundingVolumeData* bounding_volumes, 
        ldplab::Mat3* w2p_transformation, 
        ldplab::Vec3* w2p_translation, 
        size_t num_particles)
{
    using namespace ldplab;
    using namespace ldplab::rtscuda;
    unsigned int ri = blockIdx.x * blockDim.x + threadIdx.x;
    if (ri >= num_rays_per_batch)
        return;

    // Check if the ray already is in a particle space or is invalid
    if (ray_index_buffer[ri] < static_cast<int32_t>(num_particles))
        return;
    double dist;
    double min_dist = -1.0;
    int32_t min_idx = -1;

    Vec3 ray_origin = ray_origin_buffer[ri];
    Vec3 ray_direction = ray_direction_buffer[ri];

    // Check each bounding volume sequentially for intersections
    for (int32_t i = 0; i < static_cast<int32_t>(num_particles); ++i)
    {
        // Transform into particle space
        Vec3 pspace_origin = w2p_transformation[i] *
            (ray_origin + w2p_translation[i]);
        Vec3 pspace_direction = glm::normalize(
            w2p_transformation[i] * ray_direction);
        if (bounding_volumes->intersect_ray_bounding_volume(
            pspace_origin,
            pspace_direction,
            bounding_volumes->data,
            dist))
        {
            if (dist < min_dist &&
                dist > ray_min_bv_dist_buffer[ri])
            {
                min_dist = dist;
                min_idx = i;
            }
        }
    }
    // Check if the ray hits a particle bounding sphere
    if (min_idx >= 0)
    {
        // Ray hits particle with index min_idx
        ray_index_buffer[ri] = min_idx;
        ray_min_bv_dist_buffer[ri] = min_dist;
        // Transform ray from world to particle space
        ray_origin_buffer[ri] = w2p_transformation[min_idx] *
            (ray_origin + w2p_translation[min_idx]);
        ray_direction_buffer[ri] = glm::normalize(
            w2p_transformation[min_idx] * ray_direction);
    }
    else
    {
        // Ray exits the scene
        ray_index_buffer[ri] = -1;
    }
}

#endif