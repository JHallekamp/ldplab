
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "PipelineBoundingVolumeIntersection.hpp"

#include "Context.hpp"
#include <LDPLAB/Constants.hpp>

std::shared_ptr<ldplab::rtscuda::IPipelineBoundingVolumeIntersectionStage> 
    ldplab::rtscuda::IPipelineBoundingVolumeIntersectionStage::createInstance(
        const RayTracingStepCUDAInfo& info, Context& context)
{
    // Currently just create the bruteforce stage
    return std::make_shared<PipelineBoundingVolumeIntersectionBruteforce>(
        context);
}

namespace bruteforce_cuda
{
    /** @brief Bounding volume intersection device kernel. */
    __global__ void bvIntersectionKernel(
        int32_t* ray_index_buffer,
        ldplab::Vec3* ray_origin_buffer,
        ldplab::Vec3* ray_direction_buffer,
        double* ray_min_bv_dist_buffer,
        size_t num_rays_per_batch,
        ldplab::rtscuda::GenericBoundingVolumeData* bounding_volumes,
        ldplab::Mat3* w2p_transformation,
        ldplab::Vec3* w2p_translation,
        size_t num_particles);
    /** @brief Execution kernel. */
    __device__ void executeKernel(
        ldplab::rtscuda::DevicePipelineResources& resources,
        size_t ray_buffer_index);
    /** @brief Device function pointer to the actual kernel. */
    __device__ ldplab::rtscuda::pipelineExecuteBoundingVolumeIntersectionStage_t
        execution_kernel_ptr = executeKernel;
}

ldplab::rtscuda::PipelineBoundingVolumeIntersectionBruteforce::
    PipelineBoundingVolumeIntersectionBruteforce(Context& context)
    :
    m_context{ context }
{ }

void ldplab::rtscuda::PipelineBoundingVolumeIntersectionBruteforce::execute(
    size_t ray_buffer_index)
{
    //const size_t block_size = m_context.parameters.num_threads_per_block;
    //const size_t grid_size = m_context.parameters.num_rays_per_batch / block_size;
    const KernelLaunchParameter lp = getLaunchParameter();
    bruteforce_cuda::bvIntersectionKernel<<<lp.grid_size, lp.block_size>>>(
        m_context.resources.ray_buffer.index_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.origin_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.direction_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.min_bv_dist_buffers[ray_buffer_index].get(),
        m_context.parameters.num_rays_per_batch,
        m_context.resources.bounding_volumes.bounding_volume_per_particle.get(),
        m_context.resources.transformations.w2p_transformation.get(),
        m_context.resources.transformations.w2p_translation.get(),
        m_context.parameters.num_particles);
}

__device__ void bruteforce_cuda::executeKernel(
    ldplab::rtscuda::DevicePipelineResources& resources, 
    size_t ray_buffer_index)
{
    const dim3 grid_sz = resources.launch_params.boundingVolumeIntersection.grid_size;
    const dim3 block_sz = resources.launch_params.boundingVolumeIntersection.block_size;
    const unsigned int mem_sz = resources.launch_params.boundingVolumeIntersection.shared_memory_size;
    bvIntersectionKernel<<<grid_sz, block_sz, mem_sz>>>(
        resources.ray_buffer.indices[ray_buffer_index],
        resources.ray_buffer.origins[ray_buffer_index],
        resources.ray_buffer.directions[ray_buffer_index],
        resources.ray_buffer.min_bv_dists[ray_buffer_index],
        resources.parameters.num_rays_per_batch,
        resources.bounding_volumes.per_particle,
        resources.transformations.w2p_transformation,
        resources.transformations.w2p_translation,
        resources.parameters.num_particles);
}

ldplab::rtscuda::pipelineExecuteBoundingVolumeIntersectionStage_t
    ldplab::rtscuda::PipelineBoundingVolumeIntersectionBruteforce::getKernel()
{
    // Copy the function pointer to the host
    pipelineExecuteBoundingVolumeIntersectionStage_t kernel = nullptr;
    if (cudaMemcpyFromSymbol(
        &kernel,
        bruteforce_cuda::execution_kernel_ptr,
        sizeof(bruteforce_cuda::execution_kernel_ptr))
        != cudaSuccess)
        return nullptr;
    return kernel;
}

ldplab::rtscuda::KernelLaunchParameter 
    ldplab::rtscuda::PipelineBoundingVolumeIntersectionBruteforce::
        getLaunchParameter()
{
    KernelLaunchParameter p;
    p.block_size.x = 128; //m_context.device_properties.max_num_threads_per_block;
    p.grid_size.x = m_context.parameters.num_rays_per_batch / p.block_size.x +
        (m_context.parameters.num_rays_per_batch % p.block_size.x ? 1 : 0);
    return p;
}

__global__ void bruteforce_cuda::bvIntersectionKernel(
        int32_t* ray_index_buffer, 
        ldplab::Vec3* ray_origin_buffer, 
        ldplab::Vec3* ray_direction_buffer, 
        double* ray_min_bv_dist_buffer, 
        size_t num_rays_per_batch,
        ldplab::rtscuda::GenericBoundingVolumeData* bounding_volumes, 
        ldplab::Mat3* w2p_transformation, 
        ldplab::Vec3* w2p_translation, 
        size_t num_particles)
{
    using namespace ldplab;
    using namespace ldplab::rtscuda;
    unsigned int ri = blockIdx.x * blockDim.x + threadIdx.x;
    if (ri >= num_rays_per_batch)
        return;

    // Check if the ray already is in a particle space or is invalid
    if (ray_index_buffer[ri] < static_cast<int32_t>(num_particles))
        return;
    double dist;
    double min_dist = -1.0;
    int32_t min_idx = -1;

    Vec3 ray_origin = ray_origin_buffer[ri];
    Vec3 ray_direction = ray_direction_buffer[ri];

    // Check each bounding volume sequentially for intersections
    for (size_t i = 0; i < num_particles; ++i)
    {
        // Transform into particle space
        Vec3 pspace_origin = w2p_transformation[i] *
            (ray_origin + w2p_translation[i]);
        Vec3 pspace_direction = glm::normalize(
            w2p_transformation[i] * ray_direction);
        if (bounding_volumes->intersect_ray_bounding_volume(
            pspace_origin,
            pspace_direction,
            bounding_volumes->data,
            dist))
        {
            if ((dist < min_dist || min_dist < 0) &&
                dist > ray_min_bv_dist_buffer[ri])
            {
                min_dist = dist;
                min_idx = i;
            }
        }
    }
    // Check if the ray hits a particle bounding sphere
    if (min_idx >= 0)
    {
        // Ray hits particle with index min_idx
        ray_index_buffer[ri] = min_idx;
        ray_min_bv_dist_buffer[ri] = min_dist + 
            constant::intersection_tests::epsilon;
        // Transform ray from world to particle space
        ray_origin_buffer[ri] = w2p_transformation[min_idx] *
            (ray_origin + w2p_translation[min_idx]);
        ray_direction_buffer[ri] = glm::normalize(
            w2p_transformation[min_idx] * ray_direction);
    }
    else
    {
        // Ray exits the scene
        ray_index_buffer[ri] = -1;
    }
}

#endif