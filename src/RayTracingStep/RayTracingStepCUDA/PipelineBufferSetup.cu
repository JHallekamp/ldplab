
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "PipelineBufferSetup.hpp"

void ldplab::rtscuda::PipelineBufferSetup::execute()
{
    // Execute kernel
    const size_t block_size = 128;
    const size_t grid_size = m_context.parameters.num_rays_per_buffer / block_size;
    bufferSetupKernel <<<grid_size, block_size>>> (
        m_context.resources.intersection_buffer.intersection_particle_index_buffer.get(),
        m_context.resources.output_buffer.force_per_ray.get(),
        m_context.resources.output_buffer.torque_per_ray.get(),
        m_context.parameters.num_rays_per_buffer);
}

__global__ void ldplab::rtscuda::PipelineBufferSetup::bufferSetupKernel(
    int32_t* intersection_particle_index_buffer, 
    Vec3* output_force_per_ray, 
    Vec3* output_torque_per_ray, 
    size_t num_rays_per_buffer)
{
    int gi = blockIdx.x * blockDim.x + threadIdx.x;
    if (gi >= num_rays_per_buffer)
        return;
    intersection_particle_index_buffer[gi] = -1;
    output_force_per_ray[gi] = Vec3(0, 0, 0);
    output_torque_per_ray[gi] = Vec3(0, 0, 0);
}

#endif