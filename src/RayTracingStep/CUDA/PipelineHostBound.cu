
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "PipelineHostBound.hpp"

#include <array>
#include <functional>

#include "../../Utils/Log.hpp"
#include "../../Utils/Assert.hpp"
#include "../../Utils/Profiler.hpp"
#include "StageBufferSetup.hpp"
#include "StageGatherOutput.hpp"
#include "StageRayBufferReduce.hpp"

#include <Debug.hpp>

class JobWrapper : public ldplab::utils::ThreadPool::IJob
{
public:
	JobWrapper(std::function<void(size_t)> job) : m_job{ job } { }
	void execute(
        size_t job_id, 
        size_t batch_size, 
        size_t thread_id) override 
    { m_job(thread_id); }
private:
	std::function<void(size_t)> m_job;
};

ldplab::rtscuda::PipelineHostBound::PipelineHostBound(
	std::shared_ptr<utils::ThreadPool> thread_pool)
	:
	m_thread_pool{ thread_pool }
{ }

void ldplab::rtscuda::PipelineHostBound::execute()
{
    using namespace std::placeholders;
	const static std::shared_ptr<JobWrapper> job = 
		std::make_shared<JobWrapper>(
			std::bind(&PipelineHostBound::createBatchJob, this, _1));
	m_thread_pool->executeJobBatch(
		job, 
		m_context->simulation_parameter.num_parallel_batches);
}

void ldplab::rtscuda::PipelineHostBound::createBatchJob(size_t process_id)
{
	// Get batch data
	BatchData& batch_data = m_context->batch_data[process_id];
    PipelineData& pipeline_data = m_pipeline_data[process_id];

    // Initial buffer setup
    LDPLAB_PROFILING_START(pipeline_execute_step_buffer_setup);
    BufferSetup::executeStepSetup(*m_context, batch_data, pipeline_data);
    LDPLAB_PROFILING_STOP(pipeline_execute_step_buffer_setup);

    constexpr size_t initial_batch_buffer_index = 0;
    bool batches_left = false;
    size_t num_batches = 0;
    do
    {
        LDPLAB_PROFILING_START(pipeline_create_batch);
		batches_left = m_stage_is->execute(
			*m_context, 
			batch_data, 
			initial_batch_buffer_index);
        LDPLAB_PROFILING_STOP(pipeline_create_batch);
        LDPLAB_PROFILING_START(pipeline_setup_batch);
        setupBatch(batch_data);
        LDPLAB_PROFILING_STOP(pipeline_setup_batch);
        LDPLAB_PROFILING_START(pipeline_execute_batch);
        executeBatch(
            batch_data,
            pipeline_data,
			num_batches, 
			0, 
			initial_batch_buffer_index, 
			false);
        LDPLAB_PROFILING_STOP(pipeline_execute_batch);
        ++num_batches;
    } while (batches_left);
}

void ldplab::rtscuda::PipelineHostBound::setupBatch(BatchData& batch_data)
{
	m_stage_bvi->batchSetup(*m_context, batch_data);
	m_stage_is->batchSetup(*m_context, batch_data);
	m_stage_ipp->batchSetup(*m_context, batch_data);
	m_stage_pi->batchSetup(*m_context, batch_data);
	m_stage_si->batchSetup(*m_context, batch_data);
}

void ldplab::rtscuda::PipelineHostBound::executeBatch(
	BatchData& batch_data, 
    PipelineData& pipeline_data,
	size_t batch_no, 
	size_t depth, 
	size_t ray_buffer_index, 
	bool inside_particle)
{
    DebugContext dbx("logs/cuda_debug.log", [&]() {
        std::vector<std::string> out;
        std::stringstream ss;
        ss << "buffer[" << ray_buffer_index << "] | depth = " << depth <<
            " | inner particle rays = " << (inside_particle ? "true" : "false");
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "intersection_buffer[" << ray_buffer_index << "]";
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "output_ray_buffer[" << ray_buffer_index + 1 <<
            "] | depth = " << depth + 1;
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "batch_no = " << batch_no;
        out.push_back(ss.str());
        return out;
        });

    // Check if buffer contains rays
    LDPLAB_PROFILING_START(pipeline_ray_buffer_reduce);
    PipelineData::RayBufferReductionResult ray_state_count;
    ray_state_count = RayBufferReduce::execute(
        *m_context,
        batch_data, 
        pipeline_data, 
        ray_buffer_index);
    LDPLAB_PROFILING_STOP(pipeline_ray_buffer_reduce);

    if (ray_state_count.num_active_rays == 0)
        return;

    // Prepare buffer
    LDPLAB_PROFILING_START(pipeline_execute_layer_buffer_setup);
    BufferSetup::executeLayerSetup(
        *m_context,
        batch_data, 
        pipeline_data, 
        ray_buffer_index,
        ray_buffer_index);
    LDPLAB_PROFILING_STOP(pipeline_execute_layer_buffer_setup);

    batch_data.ray_data_buffers.origin_buffers.download(0, ray_buffer_index);
    batch_data.ray_data_buffers.direction_buffers.download(0, ray_buffer_index);
    batch_data.ray_data_buffers.particle_index_buffers.download(0, ray_buffer_index);
    batch_data.ray_data_buffers.intensity_buffers.download(0, ray_buffer_index);
    batch_data.intersection_data_buffers.normal_buffers.download(0, ray_buffer_index);
    batch_data.intersection_data_buffers.particle_index_buffers.download(0, ray_buffer_index);
    batch_data.intersection_data_buffers.point_buffers.download(0, ray_buffer_index);
    batch_data.output_data_buffers.force_per_particle_buffer.download();
    batch_data.output_data_buffers.force_per_ray_buffer.download(0, ray_buffer_index);
    batch_data.output_data_buffers.torque_per_particle_buffer.download();
    batch_data.output_data_buffers.torque_per_ray_buffer.download(0, ray_buffer_index);
    dbx.write([&]() {
        std::vector<std::string> out;
        out.push_back("BEFORE RAY HANDLING");
        std::stringstream ss;
        ss << "active rays: " << ray_state_count.num_active_rays;
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "world space rays: " << ray_state_count.num_world_space_rays;
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "output force: (" << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].x << ", "
            << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].y << ", "
            << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].z << ")";
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "output torque: (" << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].x << ", "
            << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].y << ", "
            << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].z << ")";
        out.push_back(ss.str());

        for (size_t i = 0; i < m_context->simulation_parameter.num_rays_per_batch; ++i)
        {
            Vec3* ray_origin = batch_data.ray_data_buffers.origin_buffers.getHostBuffer(0);
            Vec3* ray_direction = batch_data.ray_data_buffers.direction_buffers.getHostBuffer(0);
            double* intensity = batch_data.ray_data_buffers.intensity_buffers.getHostBuffer(0);
            int32_t* index = batch_data.ray_data_buffers.particle_index_buffers.getHostBuffer(0);

            ss.str(std::string());
            ss << "    ray[" << i << "]: pid = "
                << index[i] << " | o = ("
                << ray_origin[i].x << ", "
                << ray_origin[i].y << ", "
                << ray_origin[i].z << ") | d = ("
                << ray_direction[i].x << ", "
                << ray_direction[i].y << ", "
                << ray_direction[i].z << ") | i = "
                << intensity[i];
            out.push_back(ss.str());
        }

        for (size_t i = 0; i < m_context->simulation_parameter.num_rays_per_batch; ++i)
        {
            int32_t* pid = batch_data.intersection_data_buffers.particle_index_buffers.getHostBuffer(0);
            Vec3* point = batch_data.intersection_data_buffers.point_buffers.getHostBuffer(0);
            Vec3* normal = batch_data.intersection_data_buffers.normal_buffers.getHostBuffer(0);

            ss.str(std::string());
            ss << "    intersection[" << i << "]: pid = "
                << pid[i] << " | p = ("
                << point[i].x << ", "
                << point[i].y << ", "
                << point[i].z << ") | n = ("
                << normal[i].x << ", "
                << normal[i].y << ", "
                << normal[i].z << ")";
            out.push_back(ss.str());
        }
        return out;
        });

    // Switch between inside and outside particle
    if (inside_particle)
    {
        LDPLAB_PROFILING_START(pipeline_inner_particle_propagation);
        m_stage_ipp->execute(
            *m_context, 
            batch_data, 
            ray_buffer_index, 
            ray_buffer_index, 
            ray_buffer_index);
        LDPLAB_PROFILING_STOP(pipeline_inner_particle_propagation);
    }
    else
    {
        do
        {
            LDPLAB_PROFILING_START(pipeline_bounding_volume_intersection);
            m_stage_bvi->execute(
                *m_context, 
                batch_data, 
                ray_buffer_index);
            LDPLAB_PROFILING_STOP(pipeline_bounding_volume_intersection);
            LDPLAB_PROFILING_START(pipeline_particle_intersection);
            m_stage_pi->execute(
                *m_context, 
                batch_data, 
                ray_buffer_index, 
                ray_buffer_index);
            LDPLAB_PROFILING_STOP(pipeline_particle_intersection);
            LDPLAB_PROFILING_START(pipeline_ray_buffer_reduce);
            ray_state_count = RayBufferReduce::execute(
                *m_context,
                batch_data, 
                pipeline_data,
                ray_buffer_index);
            LDPLAB_PROFILING_STOP(pipeline_ray_buffer_reduce);
            if (ray_state_count.num_active_rays == 0)
                return;
        } while (ray_state_count.num_world_space_rays > 0);
    }

    // Perform surface interaction and ray branching
    constexpr std::array<bool, 2> passes{ true, false };
    for (size_t i = 0; i < passes.size(); ++i)
    {
        batch_data.ray_data_buffers.origin_buffers.download(0, ray_buffer_index);
        batch_data.ray_data_buffers.direction_buffers.download(0, ray_buffer_index);
        batch_data.ray_data_buffers.particle_index_buffers.download(0, ray_buffer_index);
        batch_data.ray_data_buffers.intensity_buffers.download(0, ray_buffer_index);
        batch_data.intersection_data_buffers.normal_buffers.download(0, ray_buffer_index);
        batch_data.intersection_data_buffers.particle_index_buffers.download(0, ray_buffer_index);
        batch_data.intersection_data_buffers.point_buffers.download(0, ray_buffer_index);
        batch_data.output_data_buffers.force_per_particle_buffer.download();
        batch_data.output_data_buffers.force_per_ray_buffer.download(0, ray_buffer_index);
        batch_data.output_data_buffers.torque_per_particle_buffer.download();
        batch_data.output_data_buffers.torque_per_ray_buffer.download(0, ray_buffer_index);
        dbx.write([&]() {
            std::vector<std::string> out;
            std::stringstream ss;
            ss << "BEFORE SURFACE INTERACTION PASS " << i;
            out.push_back(ss.str());
            ss.str(std::string());
            ss << "active rays: " << ray_state_count.num_active_rays;
            out.push_back(ss.str());
            ss.str(std::string());
            ss << "world space rays: " << ray_state_count.num_world_space_rays;
            out.push_back(ss.str());
            ss.str(std::string());
            ss << "output force: (" << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].x << ", "
                << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].y << ", "
                << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].z << ")";
            out.push_back(ss.str());
            ss.str(std::string());
            ss << "output torque: (" << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].x << ", "
                << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].y << ", "
                << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].z << ")";
            out.push_back(ss.str());

            for (size_t i = 0; i < m_context->simulation_parameter.num_rays_per_batch; ++i)
            {
                Vec3* ray_origin = batch_data.ray_data_buffers.origin_buffers.getHostBuffer(0);
                Vec3* ray_direction = batch_data.ray_data_buffers.direction_buffers.getHostBuffer(0);
                double* intensity = batch_data.ray_data_buffers.intensity_buffers.getHostBuffer(0);
                int32_t* index = batch_data.ray_data_buffers.particle_index_buffers.getHostBuffer(0);

                ss.str(std::string());
                ss << "    ray[" << i << "]: pid = "
                    << index[i] << " | o = ("
                    << ray_origin[i].x << ", "
                    << ray_origin[i].y << ", "
                    << ray_origin[i].z << ") | d = ("
                    << ray_direction[i].x << ", "
                    << ray_direction[i].y << ", "
                    << ray_direction[i].z << ") | i = "
                    << intensity[i];
                out.push_back(ss.str());
            }

            for (size_t i = 0; i < m_context->simulation_parameter.num_rays_per_batch; ++i)
            {
                int32_t* pid = batch_data.intersection_data_buffers.particle_index_buffers.getHostBuffer(0);
                Vec3* point = batch_data.intersection_data_buffers.point_buffers.getHostBuffer(0);
                Vec3* normal = batch_data.intersection_data_buffers.normal_buffers.getHostBuffer(0);

                ss.str(std::string());
                ss << "    intersection[" << i << "]: pid = "
                    << pid[i] << " | p = ("
                    << point[i].x << ", "
                    << point[i].y << ", "
                    << point[i].z << ") | n = ("
                    << normal[i].x << ", "
                    << normal[i].y << ", "
                    << normal[i].z << ")";
                out.push_back(ss.str());
            }
            return out;
            });

        const bool reflection_pass = passes[i];
        const size_t pass_lim = reflection_pass ?
            m_context->simulation_parameter.num_surface_interaction_reflection_passes :
            m_context->simulation_parameter.num_surface_interaction_transmission_passes;
        for (size_t j = 0; j < pass_lim; ++j)
        {
            LDPLAB_PROFILING_START(pipeline_surface_interaction);
            m_stage_si->execute(
                *m_context,
                batch_data,
                ray_buffer_index,
                ray_buffer_index + 1,
                ray_buffer_index,
                ray_buffer_index,
                m_context->simulation_parameter.intensity_cutoff,
                m_context->experimental_setup.medium_reflection_index,
                inside_particle,
                reflection_pass,
                j);
            LDPLAB_PROFILING_STOP(pipeline_surface_interaction);
            if (depth < m_context->simulation_parameter.max_branching_depth)
            {
                executeBatch(
                    batch_data, 
                    pipeline_data,
                    batch_no,
                    depth + 1, 
                    ray_buffer_index + 1, 
                    reflection_pass ? inside_particle : !inside_particle);
            }
        }
    }

    batch_data.ray_data_buffers.origin_buffers.download(0, ray_buffer_index);
    batch_data.ray_data_buffers.direction_buffers.download(0, ray_buffer_index);
    batch_data.ray_data_buffers.particle_index_buffers.download(0, ray_buffer_index);
    batch_data.ray_data_buffers.intensity_buffers.download(0, ray_buffer_index);
    batch_data.intersection_data_buffers.normal_buffers.download(0, ray_buffer_index);
    batch_data.intersection_data_buffers.particle_index_buffers.download(0, ray_buffer_index);
    batch_data.intersection_data_buffers.point_buffers.download(0, ray_buffer_index);
    batch_data.output_data_buffers.force_per_particle_buffer.download();
    batch_data.output_data_buffers.force_per_ray_buffer.download(0, ray_buffer_index);
    batch_data.output_data_buffers.torque_per_particle_buffer.download();
    batch_data.output_data_buffers.torque_per_ray_buffer.download(0, ray_buffer_index);
    dbx.write([&]() {
        std::vector<std::string> out;
        out.push_back("BEFORE GATHERING OUTPUT");
        std::stringstream ss;
        ss << "active rays: " << ray_state_count.num_active_rays;
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "world space rays: " << ray_state_count.num_world_space_rays;
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "output force: (" << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].x << ", "
            << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].y << ", "
            << batch_data.output_data_buffers.force_per_particle_buffer.getHostBuffer()[0].z << ")";
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "output torque: (" << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].x << ", "
            << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].y << ", "
            << batch_data.output_data_buffers.torque_per_particle_buffer.getHostBuffer()[0].z << ")";
        out.push_back(ss.str());

        int32_t* index = batch_data.ray_data_buffers.particle_index_buffers.getHostBuffer(0);
        Vec3* fpr = batch_data.output_data_buffers.force_per_ray_buffer.getHostBuffer(0);
        Vec3* tpr = batch_data.output_data_buffers.torque_per_ray_buffer.getHostBuffer(0);
        Vec3 fpp = Vec3(0, 0, 0);
        Vec3 tpp = Vec3(0, 0, 0);
        for (size_t i = 0; i < m_context->simulation_parameter.num_rays_per_batch; ++i)
        {
            if (index[i] == 0)
            {
                fpp += fpr[i];
                tpp += tpr[i];
            }
        }
        ss.str(std::string());
        ss << "accumulated ray output force: ("
            << fpp.x << ", "
            << fpp.y << ", "
            << fpp.z << ")";
        out.push_back(ss.str());
        ss.str(std::string());
        ss << "accumulated ray output torque: ("
            << tpp.x << ", "
            << tpp.y << ", "
            << tpp.z << ")";
        out.push_back(ss.str());

        for (size_t i = 0; i < m_context->simulation_parameter.num_rays_per_batch; ++i)
        {
            Vec3* ray_origin = batch_data.ray_data_buffers.origin_buffers.getHostBuffer(0);
            Vec3* ray_direction = batch_data.ray_data_buffers.direction_buffers.getHostBuffer(0);
            double* intensity = batch_data.ray_data_buffers.intensity_buffers.getHostBuffer(0);
            int32_t* index = batch_data.ray_data_buffers.particle_index_buffers.getHostBuffer(0);

            ss.str(std::string());
            ss << "    ray[" << i << "]: pid = "
                << index[i] << " | o = ("
                << ray_origin[i].x << ", "
                << ray_origin[i].y << ", "
                << ray_origin[i].z << ") | d = ("
                << ray_direction[i].x << ", "
                << ray_direction[i].y << ", "
                << ray_direction[i].z << ") | i = "
                << intensity[i];
            out.push_back(ss.str());
        }

        for (size_t i = 0; i < m_context->simulation_parameter.num_rays_per_batch; ++i)
        {
            int32_t* pid = batch_data.intersection_data_buffers.particle_index_buffers.getHostBuffer(0);
            Vec3* point = batch_data.intersection_data_buffers.point_buffers.getHostBuffer(0);
            Vec3* normal = batch_data.intersection_data_buffers.normal_buffers.getHostBuffer(0);

            ss.str(std::string());
            ss << "    intersection[" << i << "]: pid = "
                << pid[i] << " | p = ("
                << point[i].x << ", "
                << point[i].y << ", "
                << point[i].z << ") | n = ("
                << normal[i].x << ", "
                << normal[i].y << ", "
                << normal[i].z << ")";
            out.push_back(ss.str());
        }
        return out;
        });

    LDPLAB_PROFILING_START(pipeline_gather_output);
    GatherOutput::execute(
        *m_context,
        batch_data,
        pipeline_data,
        ray_buffer_index,
        ray_buffer_index);
    LDPLAB_PROFILING_STOP(pipeline_gather_output);
}

#endif
