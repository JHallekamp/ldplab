
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "ImplInitialStage.hpp"

namespace homogenous_light_bounding_sphere_projection
{
	using namespace ldplab;
	using namespace ldplab::rtscuda;
	__global__ void projectParticlesKernel(
		BoundingSphere* bounding_spheres,
		InitialStageHomogenousLightBoundingSphereProjection::Rect* projection_buffer,
		InitialStageHomogenousLightBoundingSphereProjection::HomogenousLightSource* light_buffer,
		size_t* temp_num_rays_buffer_ptr,
		double light_source_resolution_per_world_unit);
	__global__ void countTotalRaysKernelFirst(
		size_t* temp_num_rays_buffer_ptr,
		size_t num_blocks);
	__global__ void countTotalRaysKernelSecond(
		size_t* temp_num_rays_buffer_ptr,
		size_t* num_rays_buffer_ptr,
		size_t num_blocks,
		size_t num_rays_per_batch);
	__global__ void createBatchKernel(
		int32_t* ray_index_buffer,
		Vec3* ray_origin_buffer,
		Vec3* ray_direction_buffer,
		double* ray_intensity_buffer,
		double* ray_min_bv_dist_buffer,
		InitialStageHomogenousLightBoundingSphereProjection::Rect* projection_buffer,
		InitialStageHomogenousLightBoundingSphereProjection::HomogenousLightSource* light_buffer,
		size_t* num_rays_buffer,
		size_t num_particles,
		size_t num_light_sources,
		size_t num_rays_per_batch,
		size_t batch_no);
	__device__ size_t total_num_rays;
	__device__ size_t total_batch_count;
}

ldplab::rtscuda::InitialStageHomogenousLightBoundingSphereProjection::
	InitialStageHomogenousLightBoundingSphereProjection(
		double light_resolution_per_world_unit,
		DeviceBuffer<BoundingSphere>&& bounding_spheres, 
		DeviceBuffer<Rect>&& projection_buffer, 
		DeviceBuffer<HomogenousLightSource>&& light_source_buffer, 
		DeviceBuffer<size_t>&& num_rays_buffer, 
		DeviceBuffer<size_t>&& temp_num_rays_buffer)
	:
	m_light_resolution_per_world_unit{ light_resolution_per_world_unit },
	m_bounding_spheres{ std::move(bounding_spheres) },
	m_projection_buffer{ std::move(projection_buffer) },
	m_light_source_buffer{ std::move(light_source_buffer) },
	m_num_rays_buffer{ std::move(num_rays_buffer) },
	m_temp_num_rays_buffer{ std::move(temp_num_rays_buffer) }
{ }

void ldplab::rtscuda::InitialStageHomogenousLightBoundingSphereProjection::
	stepSetup(
		const SimulationState& simulation_state,
		GlobalData& global_data)
{
	BoundingSphere* spheres = m_bounding_spheres.getHostBuffer();
	for (size_t i = 0; i < global_data.experimental_setup.particles.size(); ++i)
	{
		// Get the particle instance for particle i using the interface mapping
		const ParticleInstance& particle_instance =
			simulation_state.particle_instances.find(
				global_data.interface_mapping.particle_index_to_uid.at(i))->second;
		// Get the bounding sphere in pspace
		spheres[i] = *static_cast<BoundingVolumeSphere*>(
			global_data.experimental_setup.particles[i].bounding_volume.get());
		// Translate bounding volume center to world space
		const auto& p2w_transformation =
			global_data.particle_data_buffers.p2w_transformation_buffer.getHostBuffer()[i];
		const auto& p2w_translation =
			global_data.particle_data_buffers.p2w_translation_buffer.getHostBuffer()[i];
		spheres[i].center = p2w_transformation * spheres[i].center + p2w_translation;
	}

	// Upload data
	m_bounding_spheres.upload();

	// Execute setup kernel
	using namespace homogenous_light_bounding_sphere_projection;
	const size_t grid_size = m_projection_buffer.bufferSize();
	const size_t block_size = m_light_source_buffer.bufferSize();
	const size_t mem_size = block_size * sizeof(size_t);
	projectParticlesKernel<<<grid_size, block_size>>> (
		m_bounding_spheres.getDeviceBuffer(),
		m_projection_buffer.getDeviceBuffer(),
		m_light_source_buffer.getDeviceBuffer(),
		m_temp_num_rays_buffer.getDeviceBuffer(),
		m_light_resolution_per_world_unit);
	countTotalRaysKernelFirst<<<grid_size, block_size, mem_size>>>(
		m_temp_num_rays_buffer.getDeviceBuffer(),
		grid_size);
	countTotalRaysKernelSecond<<<grid_size, block_size, mem_size>>> (
		m_temp_num_rays_buffer.getDeviceBuffer(),
		m_num_rays_buffer.getDeviceBuffer(),
		grid_size,
		global_data.simulation_parameter.num_rays_per_batch);

	// Download the total number of rays
	size_t total_rays;
	if (cudaMemcpyFromSymbol(
		&total_rays,
		total_num_rays,
		sizeof(total_num_rays)) != cudaSuccess)
	{
		total_rays = 0;
	}
	// Calculate the number of batches
	m_batch_ctr = 0;
	m_total_batch_count = 
		total_rays / global_data.simulation_parameter.num_rays_per_batch +
		(total_rays % global_data.simulation_parameter.num_rays_per_batch ? 1 : 0);
}

bool ldplab::rtscuda::InitialStageHomogenousLightBoundingSphereProjection::
	execute(
		const GlobalData& global_data, 
		BatchData& batch_data, 
		size_t initial_batch_buffer_index)
{
	using namespace homogenous_light_bounding_sphere_projection;
	const size_t block_size = 128;
	const size_t grid_size = 
		global_data.simulation_parameter.num_rays_per_batch / block_size +
		(global_data.simulation_parameter.num_rays_per_batch % block_size ? 1 : 0);
	size_t cur_batch = m_batch_ctr++;
	createBatchKernel<<<grid_size, block_size>>>(
		batch_data.ray_data_buffers.particle_index_buffers.getDeviceBuffer(initial_batch_buffer_index),
		batch_data.ray_data_buffers.origin_buffers.getDeviceBuffer(initial_batch_buffer_index),
		batch_data.ray_data_buffers.direction_buffers.getDeviceBuffer(initial_batch_buffer_index),
		batch_data.ray_data_buffers.intensity_buffers.getDeviceBuffer(initial_batch_buffer_index),
		batch_data.ray_data_buffers.min_bv_distance_buffers.getDeviceBuffer(initial_batch_buffer_index),
		m_projection_buffer.getDeviceBuffer(),
		m_light_source_buffer.getDeviceBuffer(),
		m_num_rays_buffer.getDeviceBuffer(),
		global_data.simulation_parameter.num_particles,
		m_light_source_buffer.bufferSize(),
		global_data.simulation_parameter.num_rays_per_batch,
		cur_batch);

	return (cur_batch + 1 < m_total_batch_count);
}

__global__ void homogenous_light_bounding_sphere_projection::
	projectParticlesKernel(
		BoundingSphere* bounding_spheres, 
		InitialStageHomogenousLightBoundingSphereProjection::Rect* projection_buffer,
		InitialStageHomogenousLightBoundingSphereProjection::HomogenousLightSource* light_buffer,
		size_t* temp_num_rays_buffer_ptr,
		double light_source_resolution_per_world_unit)
{
	const unsigned int projection_idx = threadIdx.x * blockDim.x + blockIdx.x;

	// ========================================================================
	// Part 1: Project sphere to plane
	InitialStageHomogenousLightBoundingSphereProjection::HomogenousLightSource light =
		light_buffer[threadIdx.x];
	BoundingSphere bs = bounding_spheres[blockIdx.x];

	// Assuming ray direction is always orthogonal to light plane
	const double t = glm::dot(light.ray_direction, light.origin - bs.center) /
		-glm::dot(light.ray_direction, light.ray_direction);
	InitialStageHomogenousLightBoundingSphereProjection::Rect projection;
	if (t < 0.0)
	{
		projection.x = -1;
		projection.y = -1;
		projection.height = 0;
		projection.width = 0;
	}
	else
	{
		const Vec3 cntr = bs.center - t * light.ray_direction - light.origin;
		const Vec2 projctr = Vec2{
		   glm::dot(cntr, glm::normalize(light.x_axis)),
		   glm::dot(cntr, glm::normalize(light.y_axis)) } *
		   light_source_resolution_per_world_unit;
		projection.x = static_cast<int>(
			projctr.x - bs.radius * light_source_resolution_per_world_unit);
		projection.y = static_cast<int>(
			projctr.y - bs.radius * light_source_resolution_per_world_unit);
		projection.width = static_cast<int>(ceil(2.0 * bs.radius * 
			light_source_resolution_per_world_unit));
		projection.height =static_cast<int>(ceil(2.0 * bs.radius *
			light_source_resolution_per_world_unit));
	}
	projection_buffer[projection_idx] = projection;

	// ========================================================================
	// Part 2: calculate projection size
	const unsigned int tid = threadIdx.x;
	temp_num_rays_buffer_ptr[projection_idx] =
		static_cast<size_t>(projection.width * projection.height);
}

__global__ void homogenous_light_bounding_sphere_projection::
	countTotalRaysKernelFirst(
		size_t* temp_num_rays_buffer_ptr,
		size_t num_blocks)
{
	using namespace ldplab;
	using namespace rtscuda;

	// Shared memory
	extern __shared__ size_t sbuf[];

	// ========================================================================
	// Part 1: Load initial sums
	const unsigned int tid = threadIdx.x;
	const unsigned int gid = blockIdx.x * blockDim.x + tid;
	sbuf[tid] = temp_num_rays_buffer_ptr[gid];
	__syncthreads();

	// ========================================================================
	// Part 2: Compute sums
	for (unsigned int i = 1; i < blockDim.x; ++i)
	{
		if (i == tid)
			sbuf[tid] += sbuf[tid - 1];
		__syncthreads();
	}

	// ========================================================================
	// Part 3: Write back results
	temp_num_rays_buffer_ptr[gid] = sbuf[tid];
}

__global__ void homogenous_light_bounding_sphere_projection::
	countTotalRaysKernelSecond(
		size_t* temp_num_rays_buffer_ptr,
		size_t* num_rays_buffer_ptr,
		size_t num_blocks, 
		size_t num_rays_per_batch)
{
	using namespace ldplab;
	using namespace rtscuda;

	// Shared memory
	extern __shared__ size_t sbuf[];

	// ========================================================================
	// Part 1: Load initial sums
	const unsigned int tid = threadIdx.x;
	const unsigned int gid = blockIdx.x * blockDim.x + tid;
	sbuf[tid] = temp_num_rays_buffer_ptr[gid];
	__syncthreads();

	// ========================================================================
	// Part 2: Compute sums for each block up until this one
	for (unsigned int i = 0; i < blockIdx.x; ++i)
	{
		sbuf[tid] += temp_num_rays_buffer_ptr[(i + 1) * blockDim.x - 1];
		__syncthreads();
	}

	// ========================================================================
	// Part 3: Write back results
	num_rays_buffer_ptr[gid] = sbuf[tid];
	if (blockIdx.x == num_blocks - 1 && tid + 1 == blockDim.x)
	{
		total_num_rays = sbuf[tid];
		total_batch_count = sbuf[tid] / num_rays_per_batch +
			(sbuf[tid] % num_rays_per_batch ? 1 : 0);
	}
}

__global__ void homogenous_light_bounding_sphere_projection::
	createBatchKernel(
		int32_t* ray_index_buffer, 
		Vec3* ray_origin_buffer, 
		Vec3* ray_direction_buffer, 
		double* ray_intensity_buffer,
		double* ray_min_bv_dist_buffer, 
		InitialStageHomogenousLightBoundingSphereProjection::Rect* projection_buffer,
		InitialStageHomogenousLightBoundingSphereProjection::HomogenousLightSource* light_buffer,
		size_t* num_rays_buffer,
		size_t num_particles,
		size_t num_light_sources,
		size_t num_rays_per_batch, 
		size_t batch_no)
{
	// ========================================================================
	// Part 1: Find which projection to use for this instance using binary search
	const unsigned int gid =
		batch_no * num_rays_per_batch + blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int ri = blockIdx.x * blockDim.x + threadIdx.x;
	ray_index_buffer[ri] = -1;
	if (gid >= total_num_rays)
		return;
	size_t low = 0;
	size_t high = (num_particles * num_light_sources) - 1;
	size_t proj_idx, nr;
	InitialStageHomogenousLightBoundingSphereProjection::Rect proj;
	do
	{
		proj_idx = (high - low) / 2;
		nr = num_rays_buffer[proj_idx];
		proj = projection_buffer[proj_idx];
		if (gid < nr && gid >= nr - proj.width * proj.height)
			break;
		else if (gid < nr)
			high = proj_idx;
		else
			low = proj_idx;
	} while (low < high);

	// ========================================================================
	// Part 2: Find which ray to create
	const unsigned int lid = gid - (nr - proj.width * proj.height);
	const int xid = static_cast<int>(lid) % proj.width;
	const int yid = static_cast<int>(lid) / proj.width;
	if (proj.x + xid < 0 ||
		proj.y + yid < 0)
		return;
	else
	{
		InitialStageHomogenousLightBoundingSphereProjection::HomogenousLightSource
			light = light_buffer[proj_idx % num_particles];
		ray_index_buffer[ri] = static_cast<int32_t>(num_particles);
		ray_origin_buffer[ri] = light.origin +
			static_cast<double>(proj.x + xid) * light.x_axis +
			static_cast<double>(proj.y + yid) * light.y_axis;
		ray_direction_buffer[ri] = light.ray_direction;
		ray_intensity_buffer[ri] = light.ray_intensity;
		ray_min_bv_dist_buffer[ri] = 0.0;
	}
}

#endif