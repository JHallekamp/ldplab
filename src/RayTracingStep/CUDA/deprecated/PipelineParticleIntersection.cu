
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "PipelineParticleIntersection.hpp"

#include "Context.hpp"

std::shared_ptr<ldplab::rtscuda::IPipelineParticleIntersectionStage> 
    ldplab::rtscuda::IPipelineParticleIntersectionStage::createInstance(
        const RayTracingStepCUDAInfo& info, 
        Context& context)
{
    // Currently we always create the generic particle geometry stage
    std::shared_ptr<ldplab::rtscuda::IPipelineParticleIntersectionStage> impl =
        std::make_shared<PipelineParticleIntersectionGenericParticleGeometry>(
            context);
    return impl;
}

namespace generic_particle_geometry_cuda
{
    __global__ void intersectionKernel(
        int32_t* ray_index_buffer,
        ldplab::Vec3* ray_origin_buffer,
        ldplab::Vec3* ray_direction_buffer,
        int32_t* intersection_index_buffer,
        ldplab::Vec3* intersection_point_buffer,
        ldplab::Vec3* intersection_normal_buffer,
        size_t num_rays_per_batch,
        ldplab::rtscuda::GenericParticleGeometryData* geometry_per_particle,
        ldplab::Mat3* p2w_transformation,
        ldplab::Vec3* p2w_translation,
        size_t num_particles);
    __device__ void executeKernel(
        ldplab::rtscuda::DevicePipelineResources& resources,
        size_t ray_buffer_index);
    __device__ ldplab::rtscuda::pipelineExecuteParticleIntersectionStage_t
        execution_kernel_ptr = executeKernel;
}

ldplab::rtscuda::PipelineParticleIntersectionGenericParticleGeometry::
    PipelineParticleIntersectionGenericParticleGeometry(Context& context)
    :
    m_context{ context }
{ }

ldplab::rtscuda::pipelineExecuteParticleIntersectionStage_t
    ldplab::rtscuda::PipelineParticleIntersectionGenericParticleGeometry::
        getKernel()
{
    using namespace generic_particle_geometry_cuda;
    // Copy the function pointer to the host
    pipelineExecuteParticleIntersectionStage_t kernel = nullptr;
    if (cudaMemcpyFromSymbol(
        &kernel,
        execution_kernel_ptr,
        sizeof(execution_kernel_ptr))
        != cudaSuccess)
        return nullptr;
    return kernel;
}

void ldplab::rtscuda::PipelineParticleIntersectionGenericParticleGeometry::
    execute(size_t ray_buffer_index)
{
    using namespace generic_particle_geometry_cuda;
    //const size_t block_size = m_context.parameters.num_threads_per_block;
    //const size_t grid_size = m_context.parameters.num_rays_per_batch / block_size;
    const KernelLaunchParameter lp = getLaunchParameter();
    intersectionKernel<<<lp.grid_size, lp.block_size>>>(
        m_context.resources.ray_buffer.index_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.origin_buffers[ray_buffer_index].get(),
        m_context.resources.ray_buffer.direction_buffers[ray_buffer_index].get(),
        m_context.resources.intersection_buffer.intersection_particle_index_buffer.get(),
        m_context.resources.intersection_buffer.intersection_point_buffer.get(),
        m_context.resources.intersection_buffer.intersection_normal_buffer.get(),
        m_context.parameters.num_rays_per_batch,
        m_context.resources.particles.geometry_per_particle.get(),
        m_context.resources.transformations.p2w_transformation.get(),
        m_context.resources.transformations.p2w_translation.get(),
        m_context.parameters.num_particles);
}

__device__ void generic_particle_geometry_cuda::executeKernel(
    ldplab::rtscuda::DevicePipelineResources& resources,
    size_t ray_buffer_index)
{
    const dim3 grid_sz = resources.launch_params.particleIntersection.grid_size;
    const dim3 block_sz = resources.launch_params.particleIntersection.block_size;
    const unsigned int mem_sz = resources.launch_params.particleIntersection.shared_memory_size;
    intersectionKernel<<<grid_sz, block_sz, mem_sz>>>(
        resources.ray_buffer.indices[ray_buffer_index],
        resources.ray_buffer.origins[ray_buffer_index],
        resources.ray_buffer.directions[ray_buffer_index],
        resources.intersection_buffer.isec_indices,
        resources.intersection_buffer.points,
        resources.intersection_buffer.normals,
        resources.parameters.num_rays_per_batch,
        resources.particles.geometry_per_particle,
        resources.transformations.p2w_transformation,
        resources.transformations.p2w_translation,
        resources.parameters.num_particles);
}

ldplab::rtscuda::KernelLaunchParameter 
    ldplab::rtscuda::PipelineParticleIntersectionGenericParticleGeometry::
        getLaunchParameter()
{
    KernelLaunchParameter p;
    p.block_size.x = 128; //m_context.device_properties.max_num_threads_per_block;
    p.grid_size.x = m_context.parameters.num_rays_per_batch / p.block_size.x +
        (m_context.parameters.num_rays_per_batch % p.block_size.x ? 1 : 0);
    return p;
}

__global__ void generic_particle_geometry_cuda::intersectionKernel(
        int32_t* ray_index_buffer,
        ldplab::Vec3* ray_origin_buffer,
        ldplab::Vec3* ray_direction_buffer,
        int32_t* intersection_index_buffer,
        ldplab::Vec3* intersection_point_buffer,
        ldplab::Vec3* intersection_normal_buffer,
        size_t num_rays_per_batch,
        ldplab::rtscuda::GenericParticleGeometryData* geometry_per_particle, 
        ldplab::Mat3* p2w_transformation,
        ldplab::Vec3* p2w_translation,
        size_t num_particles)
{
    using namespace ldplab;
    using namespace rtscuda;

    const unsigned int ri = blockIdx.x * blockDim.x + threadIdx.x;
    if (ri >= num_rays_per_batch)
        return;
    int32_t particle_index = ray_index_buffer[ri];
    if (particle_index < 0 || 
        particle_index >= static_cast<int32_t>(num_particles) ||
        particle_index == intersection_index_buffer[ri])
        return;

    Vec3 isec_pt, isec_norm;
    double dist;
    bool intersect_outside;
    if (geometry_per_particle[particle_index].intersect_ray_particle(
        ray_origin_buffer[ri],
        ray_direction_buffer[ri],
        geometry_per_particle[particle_index].data,
        isec_pt,
        isec_norm,
        dist,
        intersect_outside))
    {
        // Intersects particle
        intersection_index_buffer[ri] = particle_index;
        intersection_normal_buffer[ri] = isec_norm;
        intersection_point_buffer[ri] = isec_pt;
    }
    else
    {
        // No intersection => transform ray to world space
        ray_index_buffer[ri] = static_cast<int32_t>(num_particles);
        ray_origin_buffer[ri] = 
            p2w_transformation[particle_index] * ray_origin_buffer[ri] + 
            p2w_translation[particle_index];
        ray_direction_buffer[ri] = glm::normalize(
            p2w_transformation[particle_index] * ray_direction_buffer[ri]);
    }
}

#endif
