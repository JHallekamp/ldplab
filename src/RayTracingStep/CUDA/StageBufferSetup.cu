
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "StageBufferSetup.hpp"

namespace
{
    __global__ void bufferStepSetupKernel(
        ldplab::Vec3* output_force_per_particle,
        ldplab::Vec3* output_torque_per_particle,
        size_t num_particles)
    {
        const size_t pi = blockIdx.x * blockDim.x + threadIdx.x;
        if (pi >= num_particles)
            return;
        output_force_per_particle[pi] = ldplab::Vec3(0, 0, 0);
        output_torque_per_particle[pi] = ldplab::Vec3(0, 0, 0);
    }

    __global__ void bufferLayerSetupKernel(
        int32_t* intersection_particle_index_buffer,
        ldplab::Vec3* output_force_per_ray,
        ldplab::Vec3* output_torque_per_ray,
        size_t num_rays_per_batch)
    {
        const size_t ri = blockIdx.x * blockDim.x + threadIdx.x;
        if (ri >= num_rays_per_batch)
            return;
        intersection_particle_index_buffer[ri] = -1;
        output_force_per_ray[ri] = ldplab::Vec3(0, 0, 0);
        output_torque_per_ray[ri] = ldplab::Vec3(0, 0, 0);
    }
}

void ldplab::rtscuda::BufferSetup::executeStepSetup(
    const GlobalData& global_data,
    BatchData& batch_data,
    PipelineData& data)
{
    const PipelineData::KernelLaunchParameter& klp = data.buffer_setup_step_klp;
    bufferStepSetupKernel<<<klp.grid_size, klp.block_size, klp.shared_memory_size>>>(
        batch_data.output_data_buffers.force_per_particle_buffer.getDeviceBuffer(),
        batch_data.output_data_buffers.torque_per_particle_buffer.getDeviceBuffer(),
        global_data.simulation_parameter.num_particles);
}

void ldplab::rtscuda::BufferSetup::executeLayerSetup(
    const GlobalData& global_data,
    BatchData& batch_data,
    PipelineData& data,
    size_t buffer_index,
    size_t output_buffer_index)
{
    const PipelineData::KernelLaunchParameter& klp = data.buffer_setup_layer_klp;
    bufferLayerSetupKernel<<<klp.grid_size, klp.block_size, klp.shared_memory_size>>>(
        batch_data.intersection_data_buffers.particle_index_buffers.getDeviceBuffer(buffer_index),
        batch_data.output_data_buffers.force_per_ray_buffer.getDeviceBuffer(output_buffer_index),
        batch_data.output_data_buffers.torque_per_ray_buffer.getDeviceBuffer(output_buffer_index),
        global_data.simulation_parameter.num_rays_per_batch);
}

bool ldplab::rtscuda::BufferSetup::allocateData(
    const GlobalData& global_data, 
    PipelineData& data)
{
    constexpr size_t block_size = 128;
    PipelineData::KernelLaunchParameter& klp1 = data.buffer_setup_step_klp;
    klp1.block_size.x = block_size;
    klp1.grid_size.x =
        global_data.simulation_parameter.num_rays_per_batch / klp1.block_size.x +
        (global_data.simulation_parameter.num_rays_per_batch / klp1.block_size.x ? 1 : 0);
    data.buffer_setup_layer_klp = klp1;
    return true;
}

#endif