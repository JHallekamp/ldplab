
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "Factory.hpp"

#include "../../Utils/Log.hpp"

#include <LDPLAB/RayTracingStep/CUDA/Data.hpp>
#include <LDPLAB/RayTracingStep/CUDA/Factories.hpp>
#include <LDPLAB/RayTracingStep/CUDA/DefaultBoundingVolumeIntersectionFactories.hpp>
#include <LDPLAB/RayTracingStep/CUDA/DefaultGenericGeometryFactories.hpp>
#include <LDPLAB/RayTracingStep/CUDA/DefaultGenericMaterialFactories.hpp>
#include <LDPLAB/RayTracingStep/CUDA/DefaultInitialStageFactories.hpp>
#include <LDPLAB/RayTracingStep/CUDA/DefaultInnerParticlePropagationFactories.hpp>
#include <LDPLAB/RayTracingStep/CUDA/DefaultParticleIntersectionFactories.hpp>
#include <LDPLAB/RayTracingStep/CUDA/DefaultSurfaceInteractionFactories.hpp>

#include "PipelineDeviceBound.hpp"
#include "PipelineHostBound.hpp"
#include "StageBufferSetup.hpp"
#include "StageBufferPacking.hpp"
#include "StageBufferSort.hpp"
#include "StageGatherOutput.hpp"
#include "StageRayStateCounting.hpp"
#include "RayTracingStepCUDA.hpp"

std::shared_ptr<ldplab::rtscuda::RayTracingStepCUDA> 
    ldplab::rtscuda::Factory::createRTS(
        const RayTracingStepCUDAInfo& info, 
        ExperimentalSetup&& setup)
{
    PipelineConfiguration tmp_config{ };
    return createRTS(info, std::move(setup), tmp_config, true);
}

std::shared_ptr<ldplab::rtscuda::RayTracingStepCUDA> 
    ldplab::rtscuda::Factory::createRTS(
        const RayTracingStepCUDAInfo& info, 
        ExperimentalSetup&& setup, 
        PipelineConfiguration& user_configuration,
        bool allow_default_stage_overwrite_on_compability_error)
{
    // Retreive GPU context
    const int device_id = 0;
    if (cudaSetDevice(device_id) != cudaSuccess)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Failed to receive "\
            "cuda context for device %i", device_id);
        return nullptr;
    }

    // First, retrieve the default configuration
    PipelineConfiguration default_configuration;
    createDefaultConfiguration(info, setup, default_configuration);

    // Retrieve the present geometry type set
    std::set<IParticleGeometry::Type> present_geometry_types =
        std::move(getPresentGeometryTypes(setup));

    // Get present material type set
    std::set<IParticleMaterial::Type> present_material_types =
        std::move(getPresentMaterialTypes(setup));

    // Now combine user given config and default configuration
    PipelineConfiguration pipeline_configuration;
    if (!combineConfigurations(
        present_geometry_types,
        present_material_types,
        default_configuration,
        user_configuration,
        pipeline_configuration))
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Failed to create "\
            "RayTracingStepCPU");
        return nullptr;
    }

    // Create the interface mapping
    InterfaceMapping interface_mapping = createInterfaceMapping(setup);

    // Create execution model
    std::unique_ptr<SharedStepData> shared_data = std::make_unique<SharedStepData>();
    bool execution_model_creation_return = false;
    if (info.execution_model_info == nullptr)
    {
        execution_model_creation_return = shared_data->createExecutionModel(
            std::make_shared<ExecutionModelAutoConstructionInfo>(
                1, 
                ExecutionModelAutoConstructionInfo::DeviceModel::single_device));
    }
    else
    {
        execution_model_creation_return = shared_data->createExecutionModel(
            info.execution_model_info);
    }
    if (!execution_model_creation_return)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Failed to create pipeline "\
            "execution model");
        return nullptr;
    }

    // Find a viable pipeline configuration
    if (!createViableConfiguration(
        info,
        shared_data->execution_model,
        setup,
        interface_mapping,
        present_geometry_types,
        present_material_types,
        pipeline_configuration,
        default_configuration,
        user_configuration,
        allow_default_stage_overwrite_on_compability_error))
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Failed to create "\
            "RayTracingStepCPU");
        return nullptr;
    }

    // Log viable configuration
    logViableConfiguration(pipeline_configuration);

    // Declare the ray tracing step
    std::shared_ptr<RayTracingStepCUDA> rts = 
        std::make_shared<RayTracingStepCUDA>();

    // Create pipeline
    if (!createPipeline(
        info,
        std::move(interface_mapping),
        std::move(setup),
        pipeline_configuration,
        std::move(shared_data),
        rts))
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Failed to create "\
            "pipeline");
        return nullptr;
    }
    return rts;
}

void ldplab::rtscuda::Factory::createDefaultConfiguration(
    const RayTracingStepCUDAInfo& info, 
    const ExperimentalSetup& setup, 
    PipelineConfiguration& default_config)
{
    default_config.initial_stage = std::make_shared<
        default_factories::InitialStageHomogenousLightBoundingSphereProjectionFactory>(140);
    default_config.bounding_volume_intersection = std::make_shared<
        default_factories::BoundingSphereIntersectionBruteforceFactory>();
    default_config.particle_intersection = std::make_shared<
        default_factories::ParticleIntersectionFactory>();
    default_config.surface_interaction = std::make_shared<
        default_factories::SurfaceInteractionFactory>();
    RK4Parameter rk4_parameter = { 0.005 };
    default_config.inner_particle_propagation = std::make_shared<
        default_factories::InnerParticlePropagationRK4Factory>(rk4_parameter);

    default_config.generic_geometries.emplace(
        IParticleGeometry::Type::rod_particle,
        std::make_shared<default_factories::GenericGeometryRodFactory>());
    default_config.generic_geometries.emplace(
        IParticleGeometry::Type::sphere,
        std::make_shared<default_factories::GenericGeometrySphereFactory>());
    default_config.generic_materials.emplace(
        IParticleMaterial::Type::linear_one_directional,
        std::make_shared<default_factories::GenericMaterialLinearOneDirectionalFactory>());
}

std::set<ldplab::IParticleGeometry::Type> 
    ldplab::rtscuda::Factory::getPresentGeometryTypes(
        const ExperimentalSetup& setup)
{
    std::set<ldplab::IParticleGeometry::Type> types;
    for (size_t i = 0; i < setup.particles.size(); ++i)
    {
        if (types.find(setup.particles[i].geometry->type()) ==
            types.end())
        {
            types.insert(setup.particles[i].geometry->type());
        }
    }
    return types;
}

std::set<ldplab::IParticleMaterial::Type> 
    ldplab::rtscuda::Factory::getPresentMaterialTypes(
        const ExperimentalSetup& setup)
{
    std::set<ldplab::IParticleMaterial::Type> types;
    for (size_t i = 0; i < setup.particles.size(); ++i)
    {
        if (types.find(setup.particles[i].material->type()) ==
            types.end())
        {
            types.insert(setup.particles[i].material->type());
        }
    }
    return types;
}

bool ldplab::rtscuda::Factory::combineConfigurations(
    std::set<IParticleGeometry::Type>& geometry_types, 
    std::set<IParticleMaterial::Type>& material_types,
    PipelineConfiguration& default_config, 
    PipelineConfiguration& user_config, 
    PipelineConfiguration& combination)
{
    // Add user config
    if (user_config.bounding_volume_intersection != nullptr)
        combination.bounding_volume_intersection = user_config.bounding_volume_intersection;
    if (user_config.initial_stage != nullptr)
        combination.initial_stage = user_config.initial_stage;
    if (user_config.inner_particle_propagation != nullptr)
        combination.inner_particle_propagation = user_config.inner_particle_propagation;
    if (user_config.particle_intersection != nullptr)
        combination.particle_intersection = user_config.particle_intersection;
    if (user_config.surface_interaction != nullptr)
        combination.surface_interaction = user_config.surface_interaction;
    for (auto it = user_config.generic_geometries.begin();
        it != user_config.generic_geometries.end(); 
        it++)
    {
        if (it->second != nullptr ||
            geometry_types.find(it->first) != geometry_types.end())
            combination.generic_geometries.emplace(it->first, it->second);
    }
    for (auto it = user_config.generic_materials.begin();
        it != user_config.generic_materials.end();
        it++)
    {
        if (it->second != nullptr ||
            material_types.find(it->first) != material_types.end())
            combination.generic_materials.emplace(it->first, it->second);
    }

    // Add default config for slots that aren't set by the user config
    if (combination.bounding_volume_intersection == nullptr)
        combination.bounding_volume_intersection = default_config.bounding_volume_intersection;
    if (combination.initial_stage == nullptr)
        combination.initial_stage = default_config.initial_stage;
    if (combination.inner_particle_propagation == nullptr)
        combination.inner_particle_propagation = default_config.inner_particle_propagation;
    if (combination.particle_intersection == nullptr)
        combination.particle_intersection = default_config.particle_intersection;
    if (combination.surface_interaction == nullptr)
        combination.surface_interaction = default_config.surface_interaction;
    for (auto it = default_config.generic_geometries.begin();
        it != default_config.generic_geometries.end(); 
        it++)
    {
        if (it->second != nullptr &&
            geometry_types.find(it->first) != geometry_types.end())
        {
            if (combination.generic_geometries.find(it->first) ==
                combination.generic_geometries.end())
            {
                combination.generic_geometries.emplace(it->first, it->second);
            }
        }
    }
    for (auto it = default_config.generic_materials.begin();
        it != default_config.generic_materials.end();
        it++)
    {
        if (it->second != nullptr &&
            material_types.find(it->first) != material_types.end())
        {
            if (combination.generic_materials.find(it->first) ==
                combination.generic_materials.end())
            {
                combination.generic_materials.emplace(it->first, it->second);
            }
        }
    }

    // Check if the combined configuration is fully cast
    bool error = false;
    if (combination.bounding_volume_intersection == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline configuration "\
            "is missing a suitable bounding volume interaction factory");
        error = true;
    }
    else if (combination.initial_stage == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline configuration "\
            "is missing a suitable initial stage factory");
        error = true;
    }
    else if (combination.inner_particle_propagation == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline configuration "\
            "is missing a suitable inner particle propagation factory");
        error = true;
    }
    else if (combination.particle_intersection == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline configuration "\
            "is missing a suitable particle intersection factory");
        error = true;
    }
    else if (combination.surface_interaction == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline configuration "\
            "is missing a suitable surface interaction factory");
        error = true;
    }
    for (auto types_it = geometry_types.begin(); 
        types_it != geometry_types.end(); 
        types_it++)
    {
        if (combination.generic_geometries.find(*types_it) ==
            combination.generic_geometries.end())
        {
            LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline configuration "\
                "is missing a suitable generic geometry factory for the "\
                "particle geometry type \"%s\"",
                IParticleGeometry::typeToString(*types_it));
            error = true;
        }
    }
    for (auto types_it = material_types.begin();
        types_it != material_types.end();
        types_it++)
    {
        if (combination.generic_materials.find(*types_it) ==
            combination.generic_materials.end())
        {
            LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline configuration "\
                "is missing a suitable generic material factory for the "\
                "particle material type \"%s\"",
                IParticleMaterial::typeToString(*types_it));
            error = true;
        }
    }

    return !error;
}

ldplab::rtscuda::InterfaceMapping 
    ldplab::rtscuda::Factory::createInterfaceMapping(
        const ExperimentalSetup& setup)
{
    InterfaceMapping mapping;
    for (size_t i = 0; i < setup.particles.size(); ++i)
    {
        mapping.particle_index_to_uid.emplace(i, setup.particles[i].uid);
        mapping.particle_uid_to_index.emplace(setup.particles[i].uid, i);
    }
    return mapping;
}

bool ldplab::rtscuda::Factory::createViableConfiguration(
    const RayTracingStepCUDAInfo& info,
    const ExecutionModel& execution_model,
    const ExperimentalSetup& setup, 
    const InterfaceMapping& interface_mapping, 
    std::set<IParticleGeometry::Type>& geometry_types, 
    std::set<IParticleMaterial::Type>& material_types,
    PipelineConfiguration& configuration, 
    PipelineConfiguration& default_configuration, 
    PipelineConfiguration& user_config, 
    bool allow_default_stage_overwrite_on_compability_error)
{
    // Validate first
    PipelineConfigurationBooleanState config_state =
        validateConfigurationCompability(
            info,
            setup,
            interface_mapping,
            execution_model,
            configuration);

    // Check if compatible
    bool compatible = checkForConfigurationStateUniformity(config_state, true);
    if (compatible)
        return true;

    // Swap stages to defaults, if allowed
    if (allow_default_stage_overwrite_on_compability_error)
    {
        // Begin to swap stages to default
        PipelineConfigurationBooleanState user_defined_stages =
            checkConfigurationCast(geometry_types, material_types, user_config);
        bool has_user_defined_stages =
            !checkForConfigurationStateUniformity(user_defined_stages, false);
        while (has_user_defined_stages)
        {
            // Swap stages and update user defined stages states
            if (!config_state.bounding_volume_intersection_state)
            {
                if (!user_defined_stages.bounding_volume_intersection_state)
                    break;
                else
                {
                    LDPLAB_LOG_WARNING("RTSCUDA factory: "\
                        "Swapping incompatible user defined bounding volume "\
                        "intersection \"%s\" with default \"%s\"",
                        configuration.bounding_volume_intersection->implementationName().c_str(),
                        default_configuration.bounding_volume_intersection->implementationName().c_str());
                    configuration.bounding_volume_intersection =
                        default_configuration.bounding_volume_intersection;
                    user_defined_stages.bounding_volume_intersection_state = false;
                }
            }
            if (!config_state.initial_stage_state)
            {
                if (!user_defined_stages.initial_stage_state)
                    break;
                else
                {
                    LDPLAB_LOG_WARNING("RTSCUDA factory: "\
                        "Swapping incompatible user defined initial stage "\
                        "\"%s\" with default \"%s\"",
                        configuration.initial_stage->implementationName().c_str(),
                        default_configuration.initial_stage->implementationName().c_str());
                    configuration.initial_stage = default_configuration.initial_stage;
                    user_defined_stages.initial_stage_state = false;
                }
            }
            if (!config_state.inner_particle_propagation_state)
            {
                if (!user_defined_stages.inner_particle_propagation_state)
                    break;
                else
                {
                    LDPLAB_LOG_WARNING("RTSCUDA factory: "\
                        "Swapping incompatible user defined inner particle "\
                        "propagation \"%s\" with default \"%s\"",
                        configuration.inner_particle_propagation->implementationName().c_str(),
                        default_configuration.inner_particle_propagation->implementationName().c_str());
                    configuration.inner_particle_propagation =
                        default_configuration.inner_particle_propagation;
                    user_defined_stages.inner_particle_propagation_state = false;
                }
            }
            if (!config_state.particle_intersection_state)
            {
                if (!user_defined_stages.particle_intersection_state)
                    break;
                else
                {
                    LDPLAB_LOG_WARNING("RTSCUDA factory: "\
                        "Swapping incompatible user defined particle "\
                        "intersection \"%s\" with default \"%s\"",
                        configuration.particle_intersection->implementationName().c_str(),
                        default_configuration.particle_intersection->implementationName().c_str());
                    configuration.particle_intersection =
                        default_configuration.particle_intersection;
                    user_defined_stages.particle_intersection_state = false;
                }
            }
            if (!config_state.surface_interaction_state)
            {
                if (!user_defined_stages.surface_interaction_state)
                    break;
                else
                {
                    LDPLAB_LOG_WARNING("RTSCUDA factory: "\
                        "Swapping incompatible user defined surface "\
                        "interaction \"%s\" with default \"%s\"",
                        configuration.surface_interaction->implementationName().c_str(),
                        default_configuration.surface_interaction->implementationName().c_str());
                    configuration.surface_interaction =
                        default_configuration.surface_interaction;
                    user_defined_stages.surface_interaction_state = false;
                }
            }

            for (auto cs_it = config_state.generic_geometry_state.begin(); 
                cs_it != config_state.generic_geometry_state.end(); 
                cs_it++)
            {
                auto ud_geo_state =
                    user_defined_stages.generic_geometry_state.find(cs_it->first);
                if (ud_geo_state == user_defined_stages.generic_geometry_state.end() ||
                    ud_geo_state->second == false)
                    break;
                else
                {
                    auto config_stage =
                        configuration.generic_geometries.find(cs_it->first);
                    auto default_stage =
                        default_configuration.generic_geometries.find(cs_it->first);
                    if (config_stage == configuration.generic_geometries.end() ||
                        default_stage == default_configuration.generic_geometries.end())
                        break;
                    LDPLAB_LOG_WARNING("RTSCUDA factory: "\
                        "Swapping incompatible generic geometry \"%s\" with "\
                        "default implementation \"%s\"",
                        config_stage->second->implementationName().c_str(),
                        default_stage->second->implementationName().c_str());
                    config_stage->second = default_stage->second;
                    ud_geo_state->second = false;
                }
            }

            for (auto cs_it = config_state.generic_material_state.begin();
                cs_it != config_state.generic_material_state.end();
                cs_it++)
            {
                auto ud_mat_state =
                    user_defined_stages.generic_material_state.find(cs_it->first);
                if (ud_mat_state == user_defined_stages.generic_material_state.end() ||
                    ud_mat_state->second == false)
                    break;
                else
                {
                    auto config_stage =
                        configuration.generic_materials.find(cs_it->first);
                    auto default_stage =
                        default_configuration.generic_materials.find(cs_it->first);
                    if (config_stage == configuration.generic_materials.end() ||
                        default_stage == default_configuration.generic_materials.end())
                        break;
                    LDPLAB_LOG_WARNING("RTSCUDA factory: "\
                        "Swapping incompatible generic material \"%s\" with "\
                        "default implementation \"%s\"",
                        config_stage->second->implementationName().c_str(),
                        default_stage->second->implementationName().c_str());
                    config_stage->second = default_stage->second;
                    ud_mat_state->second = false;
                }
            }

            // Check configuration
            config_state = validateConfigurationCompability(
                info,
                setup,
                interface_mapping,
                execution_model,
                configuration);

            // Check if state is compatible
            compatible =
                checkForConfigurationStateUniformity(config_state, true);
            if (compatible)
                return true;

            // Check if there are still user defined stages
            has_user_defined_stages =
                !checkForConfigurationStateUniformity(
                    user_defined_stages, false);
        }
    }

    // Print errors
    if (!config_state.bounding_volume_intersection_state)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline bounding "\
            "volume intersection stage \"%s\" is incompatible with the given "\
            "configuration or experimental setup",
            configuration.bounding_volume_intersection->implementationName().c_str());
    }
    if (!config_state.initial_stage_state)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline initial "\
            "stage \"%s\" is incompatible with the given configuration or "\
            "experimental setup",
            configuration.initial_stage->implementationName().c_str());
    }
    if (!config_state.inner_particle_propagation_state)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline inner "\
            "particle propagation stage \"%s\" is incompatible with the given "\
            "configuration or experimental setup",
            configuration.inner_particle_propagation->implementationName().c_str());
    }
    if (!config_state.particle_intersection_state)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline particle "\
            "intersection stage \"%s\" is incompatible with the given "\
            "configuration or experimental setup",
            configuration.particle_intersection->implementationName().c_str());
    }
    if (!config_state.surface_interaction_state)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: Pipeline surface "\
            "interaction stage \"%s\" is incompatible with the given "\
            "configuration or experimental setup",
            configuration.surface_interaction->implementationName().c_str());
    }
    for (auto it = config_state.generic_geometry_state.begin();
        it != config_state.generic_geometry_state.end();
        it++)
    {
        if (!it->second)
        {
            auto geo = configuration.generic_geometries.find(it->first);
            LDPLAB_LOG_ERROR("RTSCUDA factory: Generic "\
                "geometry implementation \"%s\" for geometry type \"%s\" is "\
                "incompatible with the given configuration or "\
                "experimental setup",
                geo->second->implementationName().c_str(),
                IParticleGeometry::typeToString(it->first));
        }
    }
    for (auto it = config_state.generic_material_state.begin();
        it != config_state.generic_material_state.end();
        it++)
    {
        if (!it->second)
        {
            auto geo = configuration.generic_materials.find(it->first);
            LDPLAB_LOG_ERROR("RTSCUDA factory: Generic "\
                "material implementation \"%s\" for material type \"%s\" is "\
                "incompatible with the given configuration or "\
                "experimental setup",
                geo->second->implementationName().c_str(),
                IParticleMaterial::typeToString(it->first));
        }
    }
    return false;
}

ldplab::rtscuda::Factory::PipelineConfigurationBooleanState 
    ldplab::rtscuda::Factory::validateConfigurationCompability(
        const RayTracingStepCUDAInfo& info, 
        const ExperimentalSetup& setup, 
        const InterfaceMapping& interface_mapping, 
        const ExecutionModel& execution_model,
        PipelineConfiguration& configuration)
{
    PipelineConfigurationBooleanState state;
    state.bounding_volume_intersection_state =
        configuration.bounding_volume_intersection->checkCompability(
            info,
            execution_model,
            configuration,
            setup,
            interface_mapping);
    state.initial_stage_state =
        configuration.initial_stage->checkCompability(
            info,
            execution_model,
            configuration,
            setup,
            interface_mapping);
    state.inner_particle_propagation_state =
        configuration.inner_particle_propagation->checkCompability(
            info,
            execution_model,
            configuration,
            setup,
            interface_mapping);
    state.particle_intersection_state =
        configuration.particle_intersection->checkCompability(
            info,
            execution_model,
            configuration,
            setup,
            interface_mapping);
    state.surface_interaction_state =
        configuration.surface_interaction->checkCompability(
            info,
            execution_model,
            configuration,
            setup,
            interface_mapping);
    for (auto it = configuration.generic_geometries.begin();
        it != configuration.generic_geometries.end(); 
        ++it)
    {
        state.generic_geometry_state.emplace(it->first,
            it->second->checkCompability(
                it->first,
                info,
                execution_model,
                configuration,
                setup,
                interface_mapping));
    }
    for (auto it = configuration.generic_materials.begin();
        it != configuration.generic_materials.end();
        ++it)
    {
        state.generic_material_state.emplace(it->first,
            it->second->checkCompability(
                it->first,
                info,
                execution_model,
                configuration,
                setup,
                interface_mapping));
    }
    return state;
}

ldplab::rtscuda::Factory::PipelineConfigurationBooleanState 
    ldplab::rtscuda::Factory::checkConfigurationCast(
        std::set<IParticleGeometry::Type>& geometry_types, 
        std::set<IParticleMaterial::Type>& material_types,
        PipelineConfiguration& configuration)
{
    PipelineConfigurationBooleanState state;
    state.bounding_volume_intersection_state =
        (configuration.bounding_volume_intersection != nullptr);
    state.initial_stage_state =
        (configuration.initial_stage != nullptr);
    state.inner_particle_propagation_state =
        (configuration.inner_particle_propagation != nullptr);
    state.particle_intersection_state =
        (configuration.particle_intersection != nullptr);
    state.surface_interaction_state =
        (configuration.surface_interaction != nullptr);
    for (auto it = geometry_types.begin(); it != geometry_types.end(); ++it)
    {
        auto geo = configuration.generic_geometries.find(*it);
        if (geo == configuration.generic_geometries.end())
            state.generic_geometry_state.emplace(*it, false);
        else
            state.generic_geometry_state.emplace(*it, geo->second != nullptr);
    }
    for (auto it = material_types.begin(); it != material_types.end(); ++it)
    {
        auto mat = configuration.generic_materials.find(*it);
        if (mat == configuration.generic_materials.end())
            state.generic_material_state.emplace(*it, false);
        else
            state.generic_material_state.emplace(*it, mat->second != nullptr);
    }
    return state;
}

bool ldplab::rtscuda::Factory::checkForConfigurationStateUniformity(
    const PipelineConfigurationBooleanState& configuration_state, 
    bool desired_uniform_state)
{
    if (configuration_state.bounding_volume_intersection_state != desired_uniform_state ||
        configuration_state.initial_stage_state != desired_uniform_state ||
        configuration_state.inner_particle_propagation_state != desired_uniform_state ||
        configuration_state.particle_intersection_state != desired_uniform_state ||
        configuration_state.surface_interaction_state != desired_uniform_state)
        return false;
    for (auto it = configuration_state.generic_geometry_state.begin();
        it != configuration_state.generic_geometry_state.end();
        it++)
    {
        if (it->second != desired_uniform_state)
            return false;
    }
    for (auto it = configuration_state.generic_material_state.begin();
        it != configuration_state.generic_material_state.end();
        it++)
    {
        if (it->second != desired_uniform_state)
            return false;
    }
    return true;
}

void ldplab::rtscuda::Factory::logViableConfiguration(
    PipelineConfiguration& config)
{
    LDPLAB_LOG_INFO("RTSCUDA factory: "\
        "Pipeline configuration uses initial stage \"%s\"",
        config.initial_stage->implementationName().c_str());
    LDPLAB_LOG_INFO("RTSCUDA factory: "\
        "Pipeline configuration uses bounding volume intersection stage \"%s\"",
        config.bounding_volume_intersection->implementationName().c_str());
    LDPLAB_LOG_INFO("RTSCUDA factory: "\
        "Pipeline configuration uses particle intersection stage \"%s\"",
        config.particle_intersection->implementationName().c_str());
    LDPLAB_LOG_INFO("RTSCUDA factory: "\
        "Pipeline configuration uses surface interaction stage \"%s\"",
        config.surface_interaction->implementationName().c_str());
    LDPLAB_LOG_INFO("RTSCUDA factory: "\
        "Pipeline configuration uses inner particle propagation stage \"%s\"",
        config.inner_particle_propagation->implementationName().c_str());
    auto git = config.generic_geometries.begin();
    for (; git != config.generic_geometries.end(); ++git)
    {
        LDPLAB_LOG_INFO("RTSCUDA factory: "\
            "Pipeline configuration uses generic geometry \"%s\" for "\
            "geometry type \"%s\"",
            git->second->implementationName().c_str(),
            IParticleGeometry::typeToString(git->first));
    }
    auto mit = config.generic_materials.begin();
    for (; mit != config.generic_materials.end(); ++mit)
    {
        LDPLAB_LOG_INFO("RTSCUDA factory: "\
            "Pipeline configuration uses generic material \"%s\" for "\
            "geometry type \"%s\"",
            mit->second->implementationName().c_str(),
            IParticleMaterial::typeToString(mit->first));
    }
}

bool ldplab::rtscuda::Factory::createPipeline(
    const RayTracingStepCUDAInfo& info, 
    InterfaceMapping&& interface_mapping, 
    ExperimentalSetup&& setup, 
    PipelineConfiguration& pipeline_config,
    std::unique_ptr<SharedStepData>&& shared_data,
    std::shared_ptr<RayTracingStepCUDA>& rts)
{
    if (!shared_data->allocateResources(
        info,
        pipeline_config,
        std::move(setup),
        std::move(interface_mapping)))
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: "\
            "Failed to allocate shared pipeline data");
        return false;
    }

    // Create the pipeline stage instances
    bool error = false;

    std::shared_ptr<IInitialStage> stage_is =
        pipeline_config.initial_stage->create(
            info,
            pipeline_config,
            *shared_data);
    if (stage_is == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: "\
            "Failed to create initial stage");
        error = true;
    }
    else
        stage_is->m_parent_rts_uid = rts->uid();

    std::shared_ptr<IBoundingVolumeIntersection> stage_bvi =
        pipeline_config.bounding_volume_intersection->create(
            info,
            pipeline_config,
            *shared_data);
    if (stage_bvi == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: "\
            "Failed to create bounding volume intersection");
        error = true;
    }
    else
        stage_bvi->m_parent_rts_uid = rts->uid();

    std::shared_ptr<IParticleIntersection> stage_pi =
        pipeline_config.particle_intersection->create(
            info,
            pipeline_config,
            *shared_data);
    if (stage_pi == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: "\
            "Failed to create particle intersection");
        error = true;
    }
    else
        stage_pi->m_parent_rts_uid = rts->uid();

    std::shared_ptr<ISurfaceInteraction> stage_si =
        pipeline_config.surface_interaction->create(
            info,
            pipeline_config,
            *shared_data);
    if (stage_si == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: "\
            "Failed to create surface interaction");
        error = true;
    }
    else
        stage_si->m_parent_rts_uid = rts->uid();

    std::shared_ptr<IInnerParticlePropagation> stage_ipp =
        pipeline_config.inner_particle_propagation->create(
            info,
            pipeline_config,
            *shared_data);
    if (stage_ipp == nullptr)
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: "\
            "Failed to create inner particle propagation");
        error = true;
    }
    else
        stage_ipp->m_parent_rts_uid = rts->uid();

    if (error)
        return false;

    std::unique_ptr<IPipeline> pipeline;
    if (info.host_bound_pipeline)
    {
        std::shared_ptr<utils::ThreadPool> thread_pool =
            std::make_shared<utils::ThreadPool>(
                shared_data->execution_model.stream_contexts.size());
        pipeline = std::make_unique<PipelineHostBound>(thread_pool);
    }
    else
    {
        LDPLAB_LOG_ERROR("RTSCUDA factory: "\
            "Device bound pipeline is not implemented yet.");
        return false;
    }

    // Move context and stages to the pipeline
    pipeline->m_context = std::move(shared_data);
    pipeline->m_stage_bvi = std::move(stage_bvi);
    pipeline->m_stage_is = std::move(stage_is);
    pipeline->m_stage_ipp = std::move(stage_ipp);
    pipeline->m_stage_pi = std::move(stage_pi);
    pipeline->m_stage_si = std::move(stage_si);
    
    // Allocate pipeline data
    for (size_t i = 0; 
        i < pipeline->m_context->execution_model.stream_contexts.size(); 
        ++i)
    {
        if (!pipeline->m_context->execution_model.stream_contexts[i].deviceContext().activateDevice())
            return false;
        pipeline->m_pipeline_data.emplace_back();
        if (!BufferSetup::allocateData(
            *pipeline->m_context,
            pipeline->m_pipeline_data.back()))
        {
            LDPLAB_LOG_ERROR("RTSCUDA factory: "\
                "Failed to allocate buffer setup stage pipeline data.");
            return false;
        }
        if (!BufferPacking::allocateData(
            *pipeline->m_context,
            pipeline->m_pipeline_data.back()))
        {
            LDPLAB_LOG_ERROR("RTSCUDA factory: "\
                "Failed to allocate buffer sort stage pipeline data.");
            return false;
        }
        if (!GatherOutput::allocateData(
            *pipeline->m_context,
            pipeline->m_pipeline_data.back()))
        {
            LDPLAB_LOG_ERROR("RTSCUDA factory: "\
                "Failed to allocate gather output stage pipeline data.");
            return false;
        }
        if (!RayStateCounting::allocateData(
            *pipeline->m_context,
            pipeline->m_pipeline_data.back()))
        {
            LDPLAB_LOG_ERROR("RTSCUDA factory: "\
                "Failed to allocate ray buffer reduction stage pipeline data.");
            return false;
        }
        if (!BufferSort::allocateData(
            i,
            *pipeline->m_context,
            pipeline->m_pipeline_data.back()))
        {
            LDPLAB_LOG_ERROR("RTSCUDA factory: "\
                "Failed to allocate buffer sort stage pipeline data.");
            return false;
        }
    }

    rts->m_pipeline = std::move(pipeline);
    return true;
}


#endif