
#include <hip/hip_runtime.h>
#ifdef LDPLAB_BUILD_OPTION_ENABLE_RTSCUDA
#include "ImplBoundingVolumeIntersection.hpp"

#include <LDPLAB/Constants.hpp>

#include "IntersectionTests.hpp"

namespace sphere_brutforce
{
    __global__ void bvIntersectionKernel(
        int32_t* ray_index_buffer,
        ldplab::Vec3* ray_origin_buffer,
        ldplab::Vec3* ray_direction_buffer,
        double* ray_min_bv_dist_buffer,
        size_t num_rays_per_batch,
        ldplab::rtscuda::BoundingSphere* bounding_volumes,
        ldplab::Mat3* w2p_transformation,
        ldplab::Vec3* w2p_translation,
        size_t num_particles);
}

ldplab::rtscuda::BoundingSphere::BoundingSphere(
    const BoundingVolumeSphere& cpy)
    :
    center{ cpy.center },
    radius{ cpy.radius }
{ }

ldplab::rtscuda::BoundingSphereIntersectionBruteforce::
    BoundingSphereIntersectionBruteforce(
        DeviceBuffer<BoundingSphere>&& bounding_spheres)
    :
    m_bounding_spheres{ std::move(bounding_spheres) }
{ }

void ldplab::rtscuda::BoundingSphereIntersectionBruteforce::stepSetup(
    const SimulationState& simulation_state,
    const GlobalData& global_data)
{
    BoundingSphere* spheres = m_bounding_spheres.getHostBuffer();
    for (size_t i = 0; i < global_data.experimental_setup.particles.size(); ++i)
    {
        // Get the particle instance for particle i using the interface mapping
        const ParticleInstance& particle_instance =
            simulation_state.particle_instances.find(
                global_data.interface_mapping.particle_index_to_uid.at(i))->second;
        // Get the bounding sphere in pspace
        spheres[i] = *static_cast<BoundingVolumeSphere*>(
            global_data.experimental_setup.particles[i].bounding_volume.get());
        // Translate bounding volume center to world space
        spheres[i].center += particle_instance.position;
    }

    // Upload
    m_bounding_spheres.upload();
}

size_t ldplab::rtscuda::BoundingSphereIntersectionBruteforce::execute(
    const GlobalData& global_data, 
    BatchData& batch_data, 
    size_t ray_buffer_index)
{
#error
    return size_t();
}

__global__ void sphere_brutforce::bvIntersectionKernel(
    int32_t* ray_index_buffer,
    ldplab::Vec3* ray_origin_buffer,
    ldplab::Vec3* ray_direction_buffer,
    double* ray_min_bv_dist_buffer,
    size_t num_rays_per_batch,
    ldplab::rtscuda::BoundingSphere* bounding_volumes,
    ldplab::Mat3* w2p_transformation,
    ldplab::Vec3* w2p_translation,
    size_t num_particles)
{
    using namespace ldplab;
    using namespace ldplab::rtscuda;
    unsigned int ri = blockIdx.x * blockDim.x + threadIdx.x;
    if (ri >= num_rays_per_batch)
        return;

    // Check if the ray already is in a particle space or is invalid
    if (ray_index_buffer[ri] < static_cast<int32_t>(num_particles))
        return;
    double min_dist = -1.0;
    int32_t min_idx = -1;

    Vec3 ray_origin = ray_origin_buffer[ri];
    Vec3 ray_direction = ray_direction_buffer[ri];

    // Check each bounding volume sequentially for intersections
    for (size_t i = 0; i < num_particles; ++i)
    {
        BoundingSphere bsphere = bounding_volumes[i];
        double isec_dist_min, isec_dist_max;
        if (IntersectionTest::intersectRaySphere(
            ray_origin,
            ray_direction,
            bsphere.center,
            bsphere.radius,
            isec_dist_min,
            isec_dist_max))
        {
            if (isec_dist_min < 0)
                continue;
            if ((isec_dist_max < min_dist || min_dist < 0) &&
                isec_dist_max > ray_min_bv_dist_buffer[ri])
            {
                min_dist = isec_dist_max;
                min_idx = i;
            }
        }
    }
    // Check if the ray hits a particle bounding sphere
    if (min_idx >= 0)
    {
        // Ray hits particle with index min_idx
        ray_index_buffer[ri] = min_idx;
        ray_min_bv_dist_buffer[ri] = min_dist +
            constant::intersection_tests::epsilon;
        // Transform ray from world to particle space
        ray_origin_buffer[ri] = w2p_transformation[min_idx] *
            (ray_origin + w2p_translation[min_idx]);
        ray_direction_buffer[ri] = glm::normalize(
            w2p_transformation[min_idx] * ray_direction);
    }
    else
    {
        // Ray exits the scene
        ray_index_buffer[ri] = -1;
    }
}

#endif